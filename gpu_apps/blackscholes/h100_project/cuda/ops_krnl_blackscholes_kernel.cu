#include "hip/hip_runtime.h"
// Auto-generated at 2025-08-08 15:43:35.763485 by ops-translator

__constant__ int dims_ops_krnl_blackscholes[5][1];
static int dims_ops_krnl_blackscholes_h[5][1] = {{0}};

//  =============
//  User function
//  =============
__device__ void ops_krnl_blackscholes_gpu(ACC<float> &current, const ACC<float> &next, const ACC<float> &a, const ACC<float> &b, const ACC<float> &c) {

    current(0,0) = a(0,0) * next(-1,0) + b(0,0) * next(0,0) + c(0,0) * next(1,0);

}

//  ============================
//  Cuda kernel wrapper function
//  ============================
__global__ void ops_ops_krnl_blackscholes(float* __restrict arg0, int xstride_0, int ystride_0, 
float* __restrict arg1, int xstride_1, int ystride_1, 
float* __restrict arg2, int xstride_2, int ystride_2, 
float* __restrict arg3, int xstride_3, int ystride_3, 
float* __restrict arg4, int xstride_4, int ystride_4, 
int size0, int size1) {

    int idx_y = blockDim.y * blockIdx.y + threadIdx.y;
    int idx_x = blockDim.x * blockIdx.x + threadIdx.x;

    arg0 += idx_x * xstride_0*1 + idx_y * ystride_0*1 * dims_ops_krnl_blackscholes[0][0];
    arg1 += idx_x * xstride_1*1 + idx_y * ystride_1*1 * dims_ops_krnl_blackscholes[1][0];
    arg2 += idx_x * xstride_2*1 + idx_y * ystride_2*1 * dims_ops_krnl_blackscholes[2][0];
    arg3 += idx_x * xstride_3*1 + idx_y * ystride_3*1 * dims_ops_krnl_blackscholes[3][0];
    arg4 += idx_x * xstride_4*1 + idx_y * ystride_4*1 * dims_ops_krnl_blackscholes[4][0];

    if(idx_x < size0 && idx_y < size1) {

        ACC<float> argp0(dims_ops_krnl_blackscholes[0][0], arg0);
        const ACC<float> argp1(dims_ops_krnl_blackscholes[1][0], arg1);
        const ACC<float> argp2(dims_ops_krnl_blackscholes[2][0], arg2);
        const ACC<float> argp3(dims_ops_krnl_blackscholes[3][0], arg3);
        const ACC<float> argp4(dims_ops_krnl_blackscholes[4][0], arg4);

        ops_krnl_blackscholes_gpu(argp0, argp1, argp2, argp3, argp4);

    }// End of cuda index in_range check

}// End of cuda kernel wrapper function

//  ==================
//  Host stub function
//  ==================
#ifndef OPS_LAZY
void ops_par_loop_ops_krnl_blackscholes(
    const char * name,
    ops_block block,
    int dim,
    int *range,
    ops_arg arg0,
    ops_arg arg1,
    ops_arg arg2,
    ops_arg arg3,
    ops_arg arg4
)
{ 
#else
void ops_par_loop_ops_krnl_blackscholes_execute(ops_kernel_descriptor *desc)
{
    ops_block block = desc->block;
    int dim = desc->dim;
    int *range = desc->range;
    ops_arg arg0 = desc->args[0];
    ops_arg arg1 = desc->args[1];
    ops_arg arg2 = desc->args[2];
    ops_arg arg3 = desc->args[3];
    ops_arg arg4 = desc->args[4];
#endif

//  ======
//  Timing
//  ======
    double __t1, __t2, __c1, __c2;

    ops_arg args[5];

    args[0] = arg0;
    args[1] = arg1;
    args[2] = arg2;
    args[3] = arg3;
    args[4] = arg4;

#if defined(CHECKPOINTING) && !defined(OPS_LAZY)
    if (!ops_checkpointing_before(args, 5, range, 6)) return;
#endif

    if (block->instance->OPS_diags > 1)
    {
        ops_timing_realloc(block->instance, 6, "ops_krnl_blackscholes");
        block->instance->OPS_kernels[6].count++;
        ops_timers_core(&__c1, &__t1);
    }

//  =================================================
//  compute locally allocated range for the sub-block
//  =================================================
    int start_indx[2];
    int end_indx[2];
#ifdef OPS_MPI
    int arg_idx[2];
#endif

#if defined(OPS_LAZY) || !defined(OPS_MPI)
    for (int n = 0; n < 2; n++) {
        start_indx[n] = range[2*n];
        end_indx[n]   = range[2*n+1];
    }
#else
    if (compute_ranges(args, 5, block, range, start_indx, end_indx, arg_idx) < 0) return;
#endif

    int xdim0 = args[0].dat->size[0];
    int xdim1 = args[1].dat->size[0];
    int xdim2 = args[2].dat->size[0];
    int xdim3 = args[3].dat->size[0];
    int xdim4 = args[4].dat->size[0];

    if (xdim0 != dims_ops_krnl_blackscholes_h[0][0] || xdim1 != dims_ops_krnl_blackscholes_h[1][0] || xdim2 != dims_ops_krnl_blackscholes_h[2][0] || xdim3 != dims_ops_krnl_blackscholes_h[3][0] || xdim4 != dims_ops_krnl_blackscholes_h[4][0]) {
        dims_ops_krnl_blackscholes_h[0][0] = xdim0;
        dims_ops_krnl_blackscholes_h[1][0] = xdim1;
        dims_ops_krnl_blackscholes_h[2][0] = xdim2;
        dims_ops_krnl_blackscholes_h[3][0] = xdim3;
        dims_ops_krnl_blackscholes_h[4][0] = xdim4;

        cutilSafeCall(block->instance->ostream(), hipMemcpyToSymbol(HIP_SYMBOL( dims_ops_krnl_blackscholes), dims_ops_krnl_blackscholes_h, sizeof(dims_ops_krnl_blackscholes)));
    }

    int x_size = MAX(0,end_indx[0]-start_indx[0]);
    int y_size = MAX(0,end_indx[1]-start_indx[1]);

    dim3 grid( (x_size-1)/block->instance->OPS_block_size_x + 1, (y_size-1)/block->instance->OPS_block_size_y + 1, 1);

    dim3 tblock(block->instance->OPS_block_size_x,block->instance->OPS_block_size_y,block->instance->OPS_block_size_z);

    long long int dat0 = (block->instance->OPS_soa ? args[0].dat->type_size : args[0].dat->elem_size);
    long long int dat1 = (block->instance->OPS_soa ? args[1].dat->type_size : args[1].dat->elem_size);
    long long int dat2 = (block->instance->OPS_soa ? args[2].dat->type_size : args[2].dat->elem_size);
    long long int dat3 = (block->instance->OPS_soa ? args[3].dat->type_size : args[3].dat->elem_size);
    long long int dat4 = (block->instance->OPS_soa ? args[4].dat->type_size : args[4].dat->elem_size);

    char *p_a[5];

//  =======================
//  set up initial pointers
//  =======================
    long long int base0 = args[0].dat->base_offset + dat0 * 1 * (start_indx[0] * args[0].stencil->stride[0]);
    base0 = base0 + dat0 * 
                     args[0].dat->size[0] * 
                     (start_indx[1] * args[0].stencil->stride[1]);
    p_a[0] = (char *)args[0].data_d + base0;

    long long int base1 = args[1].dat->base_offset + dat1 * 1 * (start_indx[0] * args[1].stencil->stride[0]);
    base1 = base1 + dat1 * 
                     args[1].dat->size[0] * 
                     (start_indx[1] * args[1].stencil->stride[1]);
    p_a[1] = (char *)args[1].data_d + base1;

    long long int base2 = args[2].dat->base_offset + dat2 * 1 * (start_indx[0] * args[2].stencil->stride[0]);
    base2 = base2 + dat2 * 
                     args[2].dat->size[0] * 
                     (start_indx[1] * args[2].stencil->stride[1]);
    p_a[2] = (char *)args[2].data_d + base2;

    long long int base3 = args[3].dat->base_offset + dat3 * 1 * (start_indx[0] * args[3].stencil->stride[0]);
    base3 = base3 + dat3 * 
                     args[3].dat->size[0] * 
                     (start_indx[1] * args[3].stencil->stride[1]);
    p_a[3] = (char *)args[3].data_d + base3;

    long long int base4 = args[4].dat->base_offset + dat4 * 1 * (start_indx[0] * args[4].stencil->stride[0]);
    base4 = base4 + dat4 * 
                     args[4].dat->size[0] * 
                     (start_indx[1] * args[4].stencil->stride[1]);
    p_a[4] = (char *)args[4].data_d + base4;

//  =============
//  Halo exchange
//  =============
#ifndef OPS_LAZY
    ops_H_D_exchanges_device(args, 5);
    ops_halo_exchanges(args, 5, range);
#endif

    if (block->instance->OPS_diags > 1) { 
        ops_timers_core(&__c2, &__t2);
        block->instance->OPS_kernels[6].mpi_time += __t2 - __t1;
    }

//  ==========================================================
//  ops_dat strides for offset calculation in wrapper function
//  ==========================================================
    int xstride_0, ystride_0;
    xstride_0 = args[0].stencil->stride[0];    ystride_0 = args[0].stencil->stride[1];
    int xstride_1, ystride_1;
    xstride_1 = args[1].stencil->stride[0];    ystride_1 = args[1].stencil->stride[1];
    int xstride_2, ystride_2;
    xstride_2 = args[2].stencil->stride[0];    ystride_2 = args[2].stencil->stride[1];
    int xstride_3, ystride_3;
    xstride_3 = args[3].stencil->stride[0];    ystride_3 = args[3].stencil->stride[1];
    int xstride_4, ystride_4;
    xstride_4 = args[4].stencil->stride[0];    ystride_4 = args[4].stencil->stride[1];

//  call kernel wrapper function, passing in pointers to data
    if (x_size > 0 && y_size > 0) {

        ops_ops_krnl_blackscholes<<<grid, tblock >>> (
                   (float *)p_a[0], xstride_0, ystride_0, 
                   (float *)p_a[1], xstride_1, ystride_1, 
                   (float *)p_a[2], xstride_2, ystride_2, 
                   (float *)p_a[3], xstride_3, ystride_3, 
                   (float *)p_a[4], xstride_4, ystride_4, 
                   x_size, y_size);

    }

    cutilSafeCall(block->instance->ostream(), hipGetLastError());

    if(block->instance->OPS_diags > 1) {
        cutilSafeCall(block->instance->ostream(), hipDeviceSynchronize());
        ops_timers_core(&__c1, &__t1);
        block->instance->OPS_kernels[6].time += __t1 - __t2;
    }

#ifndef OPS_LAZY
    ops_set_dirtybit_device(args, 5);
    ops_set_halo_dirtybit3(&args[0], range);
#endif

    if (block->instance->OPS_diags > 1) {
//      ====================
//      Update kernel record
//      ====================
        ops_timers_core(&__c2, &__t2);
        block->instance->OPS_kernels[6].mpi_time += __t2 - __t1;
        block->instance->OPS_kernels[6].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg0);
        block->instance->OPS_kernels[6].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg1);
        block->instance->OPS_kernels[6].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg2);
        block->instance->OPS_kernels[6].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg3);
        block->instance->OPS_kernels[6].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg4);
    }
}

#ifdef OPS_LAZY
void ops_par_loop_ops_krnl_blackscholes(
    const char * name,
    ops_block block,
    int dim,
    int * range,
    ops_arg arg0,
    ops_arg arg1,
    ops_arg arg2,
    ops_arg arg3,
    ops_arg arg4
    )
{
    ops_arg args[5];

    args[0] = arg0;
    args[1] = arg1;
    args[2] = arg2;
    args[3] = arg3;
    args[4] = arg4;

    create_kerneldesc_and_enque(name, "ops_krnl_blackscholes", args, 5, 6, dim, 1, range, block, ops_par_loop_ops_krnl_blackscholes_execute);
}
#endif
