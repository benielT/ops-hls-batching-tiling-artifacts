#include "hip/hip_runtime.h"
// Auto-generated at 2025-08-06 12:35:43.468666 by ops-translator

__constant__ int dims_ops_krnl_calc_coefficient[6][1];
static int dims_ops_krnl_calc_coefficient_h[6][1] = {{0}};

//  =============
//  User function
//  =============
__device__ void ops_krnl_calc_coefficient_gpu(ACC<float> &a, ACC<float> &b, ACC<float> &c, const float *alpha, const float *beta, const int *idx) {

    a(0,0) = 0.5 * ((*alpha) * (idx[0] + 1) * (idx[0] + 1) - (*beta) * (idx[0] + 1));
	b(0,0) = 1 - (*alpha) * (idx[0] + 1) * (idx[0] + 1) - (*beta);
	c(0,0) = 0.5 * ((*alpha) * (idx[0] + 1) * (idx[0] + 1) + (*beta) * (idx[0] + 1));

}

//  ============================
//  Cuda kernel wrapper function
//  ============================
__global__ void ops_ops_krnl_calc_coefficient(float* __restrict arg0, int xstride_0, int ystride_0, 
float* __restrict arg1, int xstride_1, int ystride_1, 
float* __restrict arg2, int xstride_2, int ystride_2, 
const float arg3, 
const float arg4, 
int arg_idx0, int arg_idx1, 
int size0, int size1) {

    int idx_y = blockDim.y * blockIdx.y + threadIdx.y;
    int idx_x = blockDim.x * blockIdx.x + threadIdx.x;

    int arg_idx[2];
    arg_idx[0] = arg_idx0+idx_x;
    arg_idx[1] = arg_idx1+idx_y;

    arg0 += idx_x * xstride_0*1 + idx_y * ystride_0*1 * dims_ops_krnl_calc_coefficient[0][0];
    arg1 += idx_x * xstride_1*1 + idx_y * ystride_1*1 * dims_ops_krnl_calc_coefficient[1][0];
    arg2 += idx_x * xstride_2*1 + idx_y * ystride_2*1 * dims_ops_krnl_calc_coefficient[2][0];

    if(idx_x < size0 && idx_y < size1) {

        ACC<float> argp0(dims_ops_krnl_calc_coefficient[0][0], arg0);
        ACC<float> argp1(dims_ops_krnl_calc_coefficient[1][0], arg1);
        ACC<float> argp2(dims_ops_krnl_calc_coefficient[2][0], arg2);

        ops_krnl_calc_coefficient_gpu(argp0, argp1, argp2, &arg3, &arg4, arg_idx);

    }// End of cuda index in_range check

}// End of cuda kernel wrapper function

//  ==================
//  Host stub function
//  ==================
#ifndef OPS_LAZY
void ops_par_loop_ops_krnl_calc_coefficient(
    const char * name,
    ops_block block,
    int dim,
    int *range,
    ops_arg arg0,
    ops_arg arg1,
    ops_arg arg2,
    ops_arg arg3,
    ops_arg arg4,
    ops_arg arg5
)
{ 
#else
void ops_par_loop_ops_krnl_calc_coefficient_execute(ops_kernel_descriptor *desc)
{
    ops_block block = desc->block;
    int dim = desc->dim;
    int *range = desc->range;
    ops_arg arg0 = desc->args[0];
    ops_arg arg1 = desc->args[1];
    ops_arg arg2 = desc->args[2];
    ops_arg arg3 = desc->args[3];
    ops_arg arg4 = desc->args[4];
    ops_arg arg5 = desc->args[5];
#endif

//  ======
//  Timing
//  ======
    double __t1, __t2, __c1, __c2;

    ops_arg args[6];

    args[0] = arg0;
    args[1] = arg1;
    args[2] = arg2;
    args[3] = arg3;
    args[4] = arg4;
    args[5] = arg5;

#if defined(CHECKPOINTING) && !defined(OPS_LAZY)
    if (!ops_checkpointing_before(args, 6, range, 5)) return;
#endif

    if (block->instance->OPS_diags > 1)
    {
        ops_timing_realloc(block->instance, 5, "ops_krnl_calc_coefficient");
        block->instance->OPS_kernels[5].count++;
        ops_timers_core(&__c1, &__t1);
    }

//  =================================================
//  compute locally allocated range for the sub-block
//  =================================================
    int start_indx[2];
    int end_indx[2];
    int arg_idx[2];

#if defined(OPS_LAZY) || !defined(OPS_MPI)
    for (int n = 0; n < 2; n++) {
        start_indx[n] = range[2*n];
        end_indx[n]   = range[2*n+1];
    }
#else
    if (compute_ranges(args, 6, block, range, start_indx, end_indx, arg_idx) < 0) return;
#endif

#if defined(OPS_MPI)
#if defined(OPS_LAZY)
    sub_block_list sb = OPS_sub_block_list[block->index];
    arg_idx[0] = sb->decomp_disp[0]+start_indx[0];
    arg_idx[1] = sb->decomp_disp[1]+start_indx[1];
#endif  // OPS_LAZY
#else   // OPS_MPI
    arg_idx[0] = start_indx[0];
    arg_idx[1] = start_indx[1];
#endif  // OPS_MPI

    int xdim0 = args[0].dat->size[0];
    int xdim1 = args[1].dat->size[0];
    int xdim2 = args[2].dat->size[0];

    if (xdim0 != dims_ops_krnl_calc_coefficient_h[0][0] || xdim1 != dims_ops_krnl_calc_coefficient_h[1][0] || xdim2 != dims_ops_krnl_calc_coefficient_h[2][0]) {
        dims_ops_krnl_calc_coefficient_h[0][0] = xdim0;
        dims_ops_krnl_calc_coefficient_h[1][0] = xdim1;
        dims_ops_krnl_calc_coefficient_h[2][0] = xdim2;

        cutilSafeCall(block->instance->ostream(), hipMemcpyToSymbol(HIP_SYMBOL( dims_ops_krnl_calc_coefficient), dims_ops_krnl_calc_coefficient_h, sizeof(dims_ops_krnl_calc_coefficient)));
    }

    int x_size = MAX(0,end_indx[0]-start_indx[0]);
    int y_size = MAX(0,end_indx[1]-start_indx[1]);

    dim3 grid( (x_size-1)/block->instance->OPS_block_size_x + 1, (y_size-1)/block->instance->OPS_block_size_y + 1, 1);

    dim3 tblock(block->instance->OPS_block_size_x,block->instance->OPS_block_size_y,block->instance->OPS_block_size_z);

    long long int dat0 = (block->instance->OPS_soa ? args[0].dat->type_size : args[0].dat->elem_size);
    long long int dat1 = (block->instance->OPS_soa ? args[1].dat->type_size : args[1].dat->elem_size);
    long long int dat2 = (block->instance->OPS_soa ? args[2].dat->type_size : args[2].dat->elem_size);

    char *p_a[6];

//  =======================
//  set up initial pointers
//  =======================
    long long int base0 = args[0].dat->base_offset + dat0 * 1 * (start_indx[0] * args[0].stencil->stride[0]);
    base0 = base0 + dat0 * 
                     args[0].dat->size[0] * 
                     (start_indx[1] * args[0].stencil->stride[1]);
    p_a[0] = (char *)args[0].data_d + base0;

    long long int base1 = args[1].dat->base_offset + dat1 * 1 * (start_indx[0] * args[1].stencil->stride[0]);
    base1 = base1 + dat1 * 
                     args[1].dat->size[0] * 
                     (start_indx[1] * args[1].stencil->stride[1]);
    p_a[1] = (char *)args[1].data_d + base1;

    long long int base2 = args[2].dat->base_offset + dat2 * 1 * (start_indx[0] * args[2].stencil->stride[0]);
    base2 = base2 + dat2 * 
                     args[2].dat->size[0] * 
                     (start_indx[1] * args[2].stencil->stride[1]);
    p_a[2] = (char *)args[2].data_d + base2;

//  =============
//  Halo exchange
//  =============
#ifndef OPS_LAZY
    ops_H_D_exchanges_device(args, 6);
    ops_halo_exchanges(args, 6, range);
#endif

    if (block->instance->OPS_diags > 1) { 
        ops_timers_core(&__c2, &__t2);
        block->instance->OPS_kernels[5].mpi_time += __t2 - __t1;
    }

//  ==========================================================
//  ops_dat strides for offset calculation in wrapper function
//  ==========================================================
    int xstride_0, ystride_0;
    xstride_0 = args[0].stencil->stride[0];    ystride_0 = args[0].stencil->stride[1];
    int xstride_1, ystride_1;
    xstride_1 = args[1].stencil->stride[0];    ystride_1 = args[1].stencil->stride[1];
    int xstride_2, ystride_2;
    xstride_2 = args[2].stencil->stride[0];    ystride_2 = args[2].stencil->stride[1];

//  call kernel wrapper function, passing in pointers to data
    if (x_size > 0 && y_size > 0) {

        ops_ops_krnl_calc_coefficient<<<grid, tblock >>> (
                   (float *)p_a[0], xstride_0, ystride_0, 
                   (float *)p_a[1], xstride_1, ystride_1, 
                   (float *)p_a[2], xstride_2, ystride_2, 
                   *(float *)arg3.data, 
                   *(float *)arg4.data, 
                   arg_idx[0], arg_idx[1], 
                   x_size, y_size);

    }

    cutilSafeCall(block->instance->ostream(), hipGetLastError());

    if(block->instance->OPS_diags > 1) {
        cutilSafeCall(block->instance->ostream(), hipDeviceSynchronize());
        ops_timers_core(&__c1, &__t1);
        block->instance->OPS_kernels[5].time += __t1 - __t2;
    }

#ifndef OPS_LAZY
    ops_set_dirtybit_device(args, 6);
    ops_set_halo_dirtybit3(&args[0], range);
    ops_set_halo_dirtybit3(&args[1], range);
    ops_set_halo_dirtybit3(&args[2], range);
#endif

    if (block->instance->OPS_diags > 1) {
//      ====================
//      Update kernel record
//      ====================
        ops_timers_core(&__c2, &__t2);
        block->instance->OPS_kernels[5].mpi_time += __t2 - __t1;
        block->instance->OPS_kernels[5].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg0);
        block->instance->OPS_kernels[5].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg1);
        block->instance->OPS_kernels[5].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg2);
    }
}

#ifdef OPS_LAZY
void ops_par_loop_ops_krnl_calc_coefficient(
    const char * name,
    ops_block block,
    int dim,
    int * range,
    ops_arg arg0,
    ops_arg arg1,
    ops_arg arg2,
    ops_arg arg3,
    ops_arg arg4,
    ops_arg arg5
    )
{
    ops_arg args[6];

    args[0] = arg0;
    args[1] = arg1;
    args[2] = arg2;
    args[3] = arg3;
    args[4] = arg4;
    args[5] = arg5;

    create_kerneldesc_and_enque(name, "ops_krnl_calc_coefficient", args, 6, 5, dim, 1, range, block, ops_par_loop_ops_krnl_calc_coefficient_execute);
}
#endif
