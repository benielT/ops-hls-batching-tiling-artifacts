#include "hip/hip_runtime.h"
// Auto-generated at 2025-08-06 12:35:43.409852 by ops-translator

__constant__ int dims_ops_krnl_copy[2][1];
static int dims_ops_krnl_copy_h[2][1] = {{0}};

//  =============
//  User function
//  =============
__device__ void ops_krnl_copy_gpu(const ACC<float> &data, ACC<float> &data_new) {

    data_new(0,0) = data(0,0);

}

//  ============================
//  Cuda kernel wrapper function
//  ============================
__global__ void ops_ops_krnl_copy(float* __restrict arg0, int xstride_0, int ystride_0, 
float* __restrict arg1, int xstride_1, int ystride_1, 
int size0, int size1) {

    int idx_y = blockDim.y * blockIdx.y + threadIdx.y;
    int idx_x = blockDim.x * blockIdx.x + threadIdx.x;

    arg0 += idx_x * xstride_0*1 + idx_y * ystride_0*1 * dims_ops_krnl_copy[0][0];
    arg1 += idx_x * xstride_1*1 + idx_y * ystride_1*1 * dims_ops_krnl_copy[1][0];

    if(idx_x < size0 && idx_y < size1) {

        const ACC<float> argp0(dims_ops_krnl_copy[0][0], arg0);
        ACC<float> argp1(dims_ops_krnl_copy[1][0], arg1);

        ops_krnl_copy_gpu(argp0, argp1);

    }// End of cuda index in_range check

}// End of cuda kernel wrapper function

//  ==================
//  Host stub function
//  ==================
#ifndef OPS_LAZY
void ops_par_loop_ops_krnl_copy(
    const char * name,
    ops_block block,
    int dim,
    int *range,
    ops_arg arg0,
    ops_arg arg1
)
{ 
#else
void ops_par_loop_ops_krnl_copy_execute(ops_kernel_descriptor *desc)
{
    ops_block block = desc->block;
    int dim = desc->dim;
    int *range = desc->range;
    ops_arg arg0 = desc->args[0];
    ops_arg arg1 = desc->args[1];
#endif

//  ======
//  Timing
//  ======
    double __t1, __t2, __c1, __c2;

    ops_arg args[2];

    args[0] = arg0;
    args[1] = arg1;

#if defined(CHECKPOINTING) && !defined(OPS_LAZY)
    if (!ops_checkpointing_before(args, 2, range, 4)) return;
#endif

    if (block->instance->OPS_diags > 1)
    {
        ops_timing_realloc(block->instance, 4, "ops_krnl_copy");
        block->instance->OPS_kernels[4].count++;
        ops_timers_core(&__c1, &__t1);
    }

//  =================================================
//  compute locally allocated range for the sub-block
//  =================================================
    int start_indx[2];
    int end_indx[2];
#ifdef OPS_MPI
    int arg_idx[2];
#endif

#if defined(OPS_LAZY) || !defined(OPS_MPI)
    for (int n = 0; n < 2; n++) {
        start_indx[n] = range[2*n];
        end_indx[n]   = range[2*n+1];
    }
#else
    if (compute_ranges(args, 2, block, range, start_indx, end_indx, arg_idx) < 0) return;
#endif

    int xdim0 = args[0].dat->size[0];
    int xdim1 = args[1].dat->size[0];

    if (xdim0 != dims_ops_krnl_copy_h[0][0] || xdim1 != dims_ops_krnl_copy_h[1][0]) {
        dims_ops_krnl_copy_h[0][0] = xdim0;
        dims_ops_krnl_copy_h[1][0] = xdim1;

        cutilSafeCall(block->instance->ostream(), hipMemcpyToSymbol(HIP_SYMBOL( dims_ops_krnl_copy), dims_ops_krnl_copy_h, sizeof(dims_ops_krnl_copy)));
    }

    int x_size = MAX(0,end_indx[0]-start_indx[0]);
    int y_size = MAX(0,end_indx[1]-start_indx[1]);

    dim3 grid( (x_size-1)/block->instance->OPS_block_size_x + 1, (y_size-1)/block->instance->OPS_block_size_y + 1, 1);

    dim3 tblock(block->instance->OPS_block_size_x,block->instance->OPS_block_size_y,block->instance->OPS_block_size_z);

    long long int dat0 = (block->instance->OPS_soa ? args[0].dat->type_size : args[0].dat->elem_size);
    long long int dat1 = (block->instance->OPS_soa ? args[1].dat->type_size : args[1].dat->elem_size);

    char *p_a[2];

//  =======================
//  set up initial pointers
//  =======================
    long long int base0 = args[0].dat->base_offset + dat0 * 1 * (start_indx[0] * args[0].stencil->stride[0]);
    base0 = base0 + dat0 * 
                     args[0].dat->size[0] * 
                     (start_indx[1] * args[0].stencil->stride[1]);
    p_a[0] = (char *)args[0].data_d + base0;

    long long int base1 = args[1].dat->base_offset + dat1 * 1 * (start_indx[0] * args[1].stencil->stride[0]);
    base1 = base1 + dat1 * 
                     args[1].dat->size[0] * 
                     (start_indx[1] * args[1].stencil->stride[1]);
    p_a[1] = (char *)args[1].data_d + base1;

//  =============
//  Halo exchange
//  =============
#ifndef OPS_LAZY
    ops_H_D_exchanges_device(args, 2);
    ops_halo_exchanges(args, 2, range);
#endif

    if (block->instance->OPS_diags > 1) { 
        ops_timers_core(&__c2, &__t2);
        block->instance->OPS_kernels[4].mpi_time += __t2 - __t1;
    }

//  ==========================================================
//  ops_dat strides for offset calculation in wrapper function
//  ==========================================================
    int xstride_0, ystride_0;
    xstride_0 = args[0].stencil->stride[0];    ystride_0 = args[0].stencil->stride[1];
    int xstride_1, ystride_1;
    xstride_1 = args[1].stencil->stride[0];    ystride_1 = args[1].stencil->stride[1];

//  call kernel wrapper function, passing in pointers to data
    if (x_size > 0 && y_size > 0) {

        ops_ops_krnl_copy<<<grid, tblock >>> (
                   (float *)p_a[0], xstride_0, ystride_0, 
                   (float *)p_a[1], xstride_1, ystride_1, 
                   x_size, y_size);

    }

    cutilSafeCall(block->instance->ostream(), hipGetLastError());

    if(block->instance->OPS_diags > 1) {
        cutilSafeCall(block->instance->ostream(), hipDeviceSynchronize());
        ops_timers_core(&__c1, &__t1);
        block->instance->OPS_kernels[4].time += __t1 - __t2;
    }

#ifndef OPS_LAZY
    ops_set_dirtybit_device(args, 2);
    ops_set_halo_dirtybit3(&args[1], range);
#endif

    if (block->instance->OPS_diags > 1) {
//      ====================
//      Update kernel record
//      ====================
        ops_timers_core(&__c2, &__t2);
        block->instance->OPS_kernels[4].mpi_time += __t2 - __t1;
        block->instance->OPS_kernels[4].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg0);
        block->instance->OPS_kernels[4].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg1);
    }
}

#ifdef OPS_LAZY
void ops_par_loop_ops_krnl_copy(
    const char * name,
    ops_block block,
    int dim,
    int * range,
    ops_arg arg0,
    ops_arg arg1
    )
{
    ops_arg args[2];

    args[0] = arg0;
    args[1] = arg1;

    create_kerneldesc_and_enque(name, "ops_krnl_copy", args, 2, 4, dim, 1, range, block, ops_par_loop_ops_krnl_copy_execute);
}
#endif
