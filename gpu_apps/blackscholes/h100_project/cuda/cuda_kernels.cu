// Auto-generated at 2025-08-08 15:43:35.824760 by ops-translator

// headers
#define OPS_2D
#define OPS_API 2
#include "ops_lib_core.h"

#ifdef OPS_MPI
#include "ops_mpi_core.h"
#include <limits>
#endif

#include "ops_cuda_rt_support.h"
#include "ops_cuda_reduction.h"

#include <hip/hip_complex.h>

//  global constants

void ops_init_backend(){}

void ops_decl_const_char(OPS_instance *instance, int dim, char const *type, int size, char *dat, char const *name) {
    ops_execute(instance);

}

// user kernel files
#include "ops_krnl_zero_init_kernel.cu"
#include "ops_krnl_const_init_kernel.cu"
#include "ops_krnl_interior_init_kernel.cu"
#include "ops_krnl_copy_kernel.cu"
#include "ops_krnl_calc_coefficient_kernel.cu"
#include "ops_krnl_blackscholes_kernel.cu"

