#include "hip/hip_runtime.h"
// Auto-generated at 2025-08-06 12:35:43.351281 by ops-translator

__constant__ int dims_ops_krnl_interior_init[4][1];
static int dims_ops_krnl_interior_init_h[4][1] = {{0}};

//  =============
//  User function
//  =============
__device__ void ops_krnl_interior_init_gpu(ACC<float> &data, const int *idx, const float *deltaS, const float *strikePrice) {

    float tmpVal = (idx[0] + 1)*(*deltaS) - (*strikePrice);
    data(0,0) = tmpVal > 0.0 ? tmpVal : 0.0;

}

//  ============================
//  Cuda kernel wrapper function
//  ============================
__global__ void ops_ops_krnl_interior_init(float* __restrict arg0, int xstride_0, int ystride_0, 
int arg_idx0, int arg_idx1, 
const float arg2, 
const float arg3, 
int size0, int size1) {

    int idx_y = blockDim.y * blockIdx.y + threadIdx.y;
    int idx_x = blockDim.x * blockIdx.x + threadIdx.x;

    int arg_idx[2];
    arg_idx[0] = arg_idx0+idx_x;
    arg_idx[1] = arg_idx1+idx_y;

    arg0 += idx_x * xstride_0*1 + idx_y * ystride_0*1 * dims_ops_krnl_interior_init[0][0];

    if(idx_x < size0 && idx_y < size1) {

        ACC<float> argp0(dims_ops_krnl_interior_init[0][0], arg0);

        ops_krnl_interior_init_gpu(argp0, arg_idx, &arg2, &arg3);

    }// End of cuda index in_range check

}// End of cuda kernel wrapper function

//  ==================
//  Host stub function
//  ==================
#ifndef OPS_LAZY
void ops_par_loop_ops_krnl_interior_init(
    const char * name,
    ops_block block,
    int dim,
    int *range,
    ops_arg arg0,
    ops_arg arg1,
    ops_arg arg2,
    ops_arg arg3
)
{ 
#else
void ops_par_loop_ops_krnl_interior_init_execute(ops_kernel_descriptor *desc)
{
    ops_block block = desc->block;
    int dim = desc->dim;
    int *range = desc->range;
    ops_arg arg0 = desc->args[0];
    ops_arg arg1 = desc->args[1];
    ops_arg arg2 = desc->args[2];
    ops_arg arg3 = desc->args[3];
#endif

//  ======
//  Timing
//  ======
    double __t1, __t2, __c1, __c2;

    ops_arg args[4];

    args[0] = arg0;
    args[1] = arg1;
    args[2] = arg2;
    args[3] = arg3;

#if defined(CHECKPOINTING) && !defined(OPS_LAZY)
    if (!ops_checkpointing_before(args, 4, range, 3)) return;
#endif

    if (block->instance->OPS_diags > 1)
    {
        ops_timing_realloc(block->instance, 3, "ops_krnl_interior_init");
        block->instance->OPS_kernels[3].count++;
        ops_timers_core(&__c1, &__t1);
    }

//  =================================================
//  compute locally allocated range for the sub-block
//  =================================================
    int start_indx[2];
    int end_indx[2];
    int arg_idx[2];

#if defined(OPS_LAZY) || !defined(OPS_MPI)
    for (int n = 0; n < 2; n++) {
        start_indx[n] = range[2*n];
        end_indx[n]   = range[2*n+1];
    }
#else
    if (compute_ranges(args, 4, block, range, start_indx, end_indx, arg_idx) < 0) return;
#endif

#if defined(OPS_MPI)
#if defined(OPS_LAZY)
    sub_block_list sb = OPS_sub_block_list[block->index];
    arg_idx[0] = sb->decomp_disp[0]+start_indx[0];
    arg_idx[1] = sb->decomp_disp[1]+start_indx[1];
#endif  // OPS_LAZY
#else   // OPS_MPI
    arg_idx[0] = start_indx[0];
    arg_idx[1] = start_indx[1];
#endif  // OPS_MPI

    int xdim0 = args[0].dat->size[0];

    if (xdim0 != dims_ops_krnl_interior_init_h[0][0]) {
        dims_ops_krnl_interior_init_h[0][0] = xdim0;

        cutilSafeCall(block->instance->ostream(), hipMemcpyToSymbol(HIP_SYMBOL( dims_ops_krnl_interior_init), dims_ops_krnl_interior_init_h, sizeof(dims_ops_krnl_interior_init)));
    }

    int x_size = MAX(0,end_indx[0]-start_indx[0]);
    int y_size = MAX(0,end_indx[1]-start_indx[1]);

    dim3 grid( (x_size-1)/block->instance->OPS_block_size_x + 1, (y_size-1)/block->instance->OPS_block_size_y + 1, 1);

    dim3 tblock(block->instance->OPS_block_size_x,block->instance->OPS_block_size_y,block->instance->OPS_block_size_z);

    long long int dat0 = (block->instance->OPS_soa ? args[0].dat->type_size : args[0].dat->elem_size);

    char *p_a[4];

//  =======================
//  set up initial pointers
//  =======================
    long long int base0 = args[0].dat->base_offset + dat0 * 1 * (start_indx[0] * args[0].stencil->stride[0]);
    base0 = base0 + dat0 * 
                     args[0].dat->size[0] * 
                     (start_indx[1] * args[0].stencil->stride[1]);
    p_a[0] = (char *)args[0].data_d + base0;

//  =============
//  Halo exchange
//  =============
#ifndef OPS_LAZY
    ops_H_D_exchanges_device(args, 4);
    ops_halo_exchanges(args, 4, range);
#endif

    if (block->instance->OPS_diags > 1) { 
        ops_timers_core(&__c2, &__t2);
        block->instance->OPS_kernels[3].mpi_time += __t2 - __t1;
    }

//  ==========================================================
//  ops_dat strides for offset calculation in wrapper function
//  ==========================================================
    int xstride_0, ystride_0;
    xstride_0 = args[0].stencil->stride[0];    ystride_0 = args[0].stencil->stride[1];

//  call kernel wrapper function, passing in pointers to data
    if (x_size > 0 && y_size > 0) {

        ops_ops_krnl_interior_init<<<grid, tblock >>> (
                   (float *)p_a[0], xstride_0, ystride_0, 
                   arg_idx[0], arg_idx[1], 
                   *(float *)arg2.data, 
                   *(float *)arg3.data, 
                   x_size, y_size);

    }

    cutilSafeCall(block->instance->ostream(), hipGetLastError());

    if(block->instance->OPS_diags > 1) {
        cutilSafeCall(block->instance->ostream(), hipDeviceSynchronize());
        ops_timers_core(&__c1, &__t1);
        block->instance->OPS_kernels[3].time += __t1 - __t2;
    }

#ifndef OPS_LAZY
    ops_set_dirtybit_device(args, 4);
    ops_set_halo_dirtybit3(&args[0], range);
#endif

    if (block->instance->OPS_diags > 1) {
//      ====================
//      Update kernel record
//      ====================
        ops_timers_core(&__c2, &__t2);
        block->instance->OPS_kernels[3].mpi_time += __t2 - __t1;
        block->instance->OPS_kernels[3].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg0);
    }
}

#ifdef OPS_LAZY
void ops_par_loop_ops_krnl_interior_init(
    const char * name,
    ops_block block,
    int dim,
    int * range,
    ops_arg arg0,
    ops_arg arg1,
    ops_arg arg2,
    ops_arg arg3
    )
{
    ops_arg args[4];

    args[0] = arg0;
    args[1] = arg1;
    args[2] = arg2;
    args[3] = arg3;

    create_kerneldesc_and_enque(name, "ops_krnl_interior_init", args, 4, 3, dim, 1, range, block, ops_par_loop_ops_krnl_interior_init_execute);
}
#endif
