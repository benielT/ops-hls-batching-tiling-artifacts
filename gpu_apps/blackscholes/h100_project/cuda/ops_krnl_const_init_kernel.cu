#include "hip/hip_runtime.h"
// Auto-generated at 2025-08-06 12:35:43.292795 by ops-translator

__constant__ int dims_ops_krnl_const_init[2][1];
static int dims_ops_krnl_const_init_h[2][1] = {{0}};

//  =============
//  User function
//  =============
__device__ void ops_krnl_const_init_gpu(ACC<float> &data, const float *val) {

    data(0,0) = *val;

}

//  ============================
//  Cuda kernel wrapper function
//  ============================
__global__ void ops_ops_krnl_const_init(float* __restrict arg0, int xstride_0, int ystride_0, 
const float arg1, 
int size0, int size1) {

    int idx_y = blockDim.y * blockIdx.y + threadIdx.y;
    int idx_x = blockDim.x * blockIdx.x + threadIdx.x;

    arg0 += idx_x * xstride_0*1 + idx_y * ystride_0*1 * dims_ops_krnl_const_init[0][0];

    if(idx_x < size0 && idx_y < size1) {

        ACC<float> argp0(dims_ops_krnl_const_init[0][0], arg0);

        ops_krnl_const_init_gpu(argp0, &arg1);

    }// End of cuda index in_range check

}// End of cuda kernel wrapper function

//  ==================
//  Host stub function
//  ==================
#ifndef OPS_LAZY
void ops_par_loop_ops_krnl_const_init(
    const char * name,
    ops_block block,
    int dim,
    int *range,
    ops_arg arg0,
    ops_arg arg1
)
{ 
#else
void ops_par_loop_ops_krnl_const_init_execute(ops_kernel_descriptor *desc)
{
    ops_block block = desc->block;
    int dim = desc->dim;
    int *range = desc->range;
    ops_arg arg0 = desc->args[0];
    ops_arg arg1 = desc->args[1];
#endif

//  ======
//  Timing
//  ======
    double __t1, __t2, __c1, __c2;

    ops_arg args[2];

    args[0] = arg0;
    args[1] = arg1;

#if defined(CHECKPOINTING) && !defined(OPS_LAZY)
    if (!ops_checkpointing_before(args, 2, range, 2)) return;
#endif

    if (block->instance->OPS_diags > 1)
    {
        ops_timing_realloc(block->instance, 2, "ops_krnl_const_init");
        block->instance->OPS_kernels[2].count++;
        ops_timers_core(&__c1, &__t1);
    }

//  =================================================
//  compute locally allocated range for the sub-block
//  =================================================
    int start_indx[2];
    int end_indx[2];
#ifdef OPS_MPI
    int arg_idx[2];
#endif

#if defined(OPS_LAZY) || !defined(OPS_MPI)
    for (int n = 0; n < 2; n++) {
        start_indx[n] = range[2*n];
        end_indx[n]   = range[2*n+1];
    }
#else
    if (compute_ranges(args, 2, block, range, start_indx, end_indx, arg_idx) < 0) return;
#endif

    int xdim0 = args[0].dat->size[0];

    if (xdim0 != dims_ops_krnl_const_init_h[0][0]) {
        dims_ops_krnl_const_init_h[0][0] = xdim0;

        cutilSafeCall(block->instance->ostream(), hipMemcpyToSymbol(HIP_SYMBOL( dims_ops_krnl_const_init), dims_ops_krnl_const_init_h, sizeof(dims_ops_krnl_const_init)));
    }

    int x_size = MAX(0,end_indx[0]-start_indx[0]);
    int y_size = MAX(0,end_indx[1]-start_indx[1]);

    dim3 grid( (x_size-1)/block->instance->OPS_block_size_x + 1, (y_size-1)/block->instance->OPS_block_size_y + 1, 1);

    dim3 tblock(block->instance->OPS_block_size_x,block->instance->OPS_block_size_y,block->instance->OPS_block_size_z);

    long long int dat0 = (block->instance->OPS_soa ? args[0].dat->type_size : args[0].dat->elem_size);

    char *p_a[2];

//  =======================
//  set up initial pointers
//  =======================
    long long int base0 = args[0].dat->base_offset + dat0 * 1 * (start_indx[0] * args[0].stencil->stride[0]);
    base0 = base0 + dat0 * 
                     args[0].dat->size[0] * 
                     (start_indx[1] * args[0].stencil->stride[1]);
    p_a[0] = (char *)args[0].data_d + base0;

//  =============
//  Halo exchange
//  =============
#ifndef OPS_LAZY
    ops_H_D_exchanges_device(args, 2);
    ops_halo_exchanges(args, 2, range);
#endif

    if (block->instance->OPS_diags > 1) { 
        ops_timers_core(&__c2, &__t2);
        block->instance->OPS_kernels[2].mpi_time += __t2 - __t1;
    }

//  ==========================================================
//  ops_dat strides for offset calculation in wrapper function
//  ==========================================================
    int xstride_0, ystride_0;
    xstride_0 = args[0].stencil->stride[0];    ystride_0 = args[0].stencil->stride[1];

//  call kernel wrapper function, passing in pointers to data
    if (x_size > 0 && y_size > 0) {

        ops_ops_krnl_const_init<<<grid, tblock >>> (
                   (float *)p_a[0], xstride_0, ystride_0, 
                   *(float *)arg1.data, 
                   x_size, y_size);

    }

    cutilSafeCall(block->instance->ostream(), hipGetLastError());

    if(block->instance->OPS_diags > 1) {
        cutilSafeCall(block->instance->ostream(), hipDeviceSynchronize());
        ops_timers_core(&__c1, &__t1);
        block->instance->OPS_kernels[2].time += __t1 - __t2;
    }

#ifndef OPS_LAZY
    ops_set_dirtybit_device(args, 2);
    ops_set_halo_dirtybit3(&args[0], range);
#endif

    if (block->instance->OPS_diags > 1) {
//      ====================
//      Update kernel record
//      ====================
        ops_timers_core(&__c2, &__t2);
        block->instance->OPS_kernels[2].mpi_time += __t2 - __t1;
        block->instance->OPS_kernels[2].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg0);
    }
}

#ifdef OPS_LAZY
void ops_par_loop_ops_krnl_const_init(
    const char * name,
    ops_block block,
    int dim,
    int * range,
    ops_arg arg0,
    ops_arg arg1
    )
{
    ops_arg args[2];

    args[0] = arg0;
    args[1] = arg1;

    create_kerneldesc_and_enque(name, "ops_krnl_const_init", args, 2, 2, dim, 1, range, block, ops_par_loop_ops_krnl_const_init_execute);
}
#endif
