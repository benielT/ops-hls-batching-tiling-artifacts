#include "hip/hip_runtime.h"
// Auto-generated at 2025-08-06 23:23:48.342388 by ops-translator

__constant__ int dims_fd3d_pml_kernel2[33][2];
static int dims_fd3d_pml_kernel2_h[33][2] = {{0}};

//  =============
//  User function
//  =============
__device__ void fd3d_pml_kernel2_gpu(const int *dispx, const int *dispy, const int *dispz, const int *idx, const float *dt, const float *scale1, const float *scale2, const ACC<float> &rho, const ACC<float> &mu, const ACC<float> &yy_0, const ACC<float> &yy_1, const ACC<float> &yy_2, const ACC<float> &yy_3, const ACC<float> &yy_4, const ACC<float> &yy_5, const ACC<float> &dyyIn_0, const ACC<float> &dyyIn_1, const ACC<float> &dyyIn_2, const ACC<float> &dyyIn_3, const ACC<float> &dyyIn_4, const ACC<float> &dyyIn_5, ACC<float> &dyyOut_0, ACC<float> &dyyOut_1, ACC<float> &dyyOut_2, ACC<float> &dyyOut_3, ACC<float> &dyyOut_4, ACC<float> &dyyOut_5, ACC<float> &sum_0, ACC<float> &sum_1, ACC<float> &sum_2, ACC<float> &sum_3, ACC<float> &sum_4, ACC<float> &sum_5) {

    const float c[9] = {0.0035714285714285713,-0.0380952380952381,0.2,-0.8,0.0,0.8,-0.2,0.0380952380952381,-0.0035714285714285713};

    int xbeg=half_order;
    int xend=nx-half_order;
    int ybeg=half_order;
    int yend=ny-half_order;
    int zbeg=half_order;
    int zend=nz-half_order;
    int xpmlbeg=xbeg+pml_width;
    int ypmlbeg=ybeg+pml_width;
    int zpmlbeg=zbeg+pml_width;
    int xpmlend=xend-pml_width;
    int ypmlend=yend-pml_width;
    int zpmlend=zend-pml_width;

    float sigma = mu(0,0,0)/rho(0,0,0);
    float sigmax=0.0;
    float sigmay=0.0;
    float sigmaz=0.0;
    if(idx[0]<=xbeg+pml_width){
        sigmax = (xbeg+pml_width-idx[0])*sigma * 0.1f;
    }
    if(idx[0]>=xend-pml_width){
        sigmax=(idx[0]-(xend-pml_width))*sigma * 0.1f;
    }
    if(idx[1]<=ybeg+pml_width){
        sigmay=(ybeg+pml_width-idx[1])*sigma * 0.1f;
    }
    if(idx[1]>=yend-pml_width){
        sigmay=(idx[1]-(yend-pml_width))*sigma * 0.1f;
    }
    if(idx[2]<=zbeg+pml_width){
        sigmaz=(zbeg+pml_width-idx[2])*sigma * 0.1f;
    }
    if(idx[2]>=zend-pml_width){
        sigmaz=(idx[2]-(zend-pml_width))*sigma * 0.1f;
    }

    float px = dyyIn_0(0,0,0);
    float py = dyyIn_1(0,0,0);
    float pz = dyyIn_2(0,0,0);

    float vx = dyyIn_3(0,0,0);
    float vy = dyyIn_4(0,0,0);
    float vz = dyyIn_5(0,0,0);

    float vxx=0.0;
    float vxy=0.0;
    float vxz=0.0;

    float vyx=0.0;
    float vyy=0.0;
    float vyz=0.0;

    float vzx=0.0;
    float vzy=0.0;
    float vzz=0.0;

    float pxx=0.0;
    float pxy=0.0;
    float pxz=0.0;

    float pyx=0.0;
    float pyy=0.0;
    float pyz=0.0;

    float pzx=0.0;
    float pzy=0.0;
    float pzz=0.0;

    pxx += dyyIn_0(-4,0,0)*c[-4+half_order];
    pyx += dyyIn_1(-4,0,0)*c[-4+half_order];
    pzx += dyyIn_2(-4,0,0)*c[-4+half_order];

    vxx += dyyIn_3(-4,0,0)*c[-4+half_order];
    vyx += dyyIn_4(-4,0,0)*c[-4+half_order];
    vzx += dyyIn_5(-4,0,0)*c[-4+half_order];

    pxy += dyyIn_0(0,-4,0)*c[-4+half_order];
    pyy += dyyIn_1(0,-4,0)*c[-4+half_order];
    pzy += dyyIn_2(0,-4,0)*c[-4+half_order];

    vxy += dyyIn_3(0,-4,0)*c[-4+half_order];
    vyy += dyyIn_4(0,-4,0)*c[-4+half_order];
    vzy += dyyIn_5(0,-4,0)*c[-4+half_order];

    pxz += dyyIn_0(0,0,-4)*c[-4+half_order];
    pyz += dyyIn_1(0,0,-4)*c[-4+half_order];
    pzz += dyyIn_2(0,0,-4)*c[-4+half_order];

    vxz += dyyIn_3(0,0,-4)*c[-4+half_order];
    vyz += dyyIn_4(0,0,-4)*c[-4+half_order];
    vzz += dyyIn_5(0,0,-4)*c[-4+half_order];

    pxx += dyyIn_0(-3,0,0)*c[-3+half_order];
    pyx += dyyIn_1(-3,0,0)*c[-3+half_order];
    pzx += dyyIn_2(-3,0,0)*c[-3+half_order];

    vxx += dyyIn_3(-3,0,0)*c[-3+half_order];
    vyx += dyyIn_4(-3,0,0)*c[-3+half_order];
    vzx += dyyIn_5(-3,0,0)*c[-3+half_order];

    pxy += dyyIn_0(0,-3,0)*c[-3+half_order];
    pyy += dyyIn_1(0,-3,0)*c[-3+half_order];
    pzy += dyyIn_2(0,-3,0)*c[-3+half_order];

    vxy += dyyIn_3(0,-3,0)*c[-3+half_order];
    vyy += dyyIn_4(0,-3,0)*c[-3+half_order];
    vzy += dyyIn_5(0,-3,0)*c[-3+half_order];

    pxz += dyyIn_0(0,0,-3)*c[-3+half_order];
    pyz += dyyIn_1(0,0,-3)*c[-3+half_order];
    pzz += dyyIn_2(0,0,-3)*c[-3+half_order];

    vxz += dyyIn_3(0,0,-3)*c[-3+half_order];
    vyz += dyyIn_4(0,0,-3)*c[-3+half_order];
    vzz += dyyIn_5(0,0,-3)*c[-3+half_order];

    pxx += dyyIn_0(-2,0,0)*c[-2+half_order];
    pyx += dyyIn_1(-2,0,0)*c[-2+half_order];
    pzx += dyyIn_2(-2,0,0)*c[-2+half_order];

    vxx += dyyIn_3(-2,0,0)*c[-2+half_order];
    vyx += dyyIn_4(-2,0,0)*c[-2+half_order];
    vzx += dyyIn_5(-2,0,0)*c[-2+half_order];

    pxy += dyyIn_0(0,-2,0)*c[-2+half_order];
    pyy += dyyIn_1(0,-2,0)*c[-2+half_order];
    pzy += dyyIn_2(0,-2,0)*c[-2+half_order];

    vxy += dyyIn_3(0,-2,0)*c[-2+half_order];
    vyy += dyyIn_4(0,-2,0)*c[-2+half_order];
    vzy += dyyIn_5(0,-2,0)*c[-2+half_order];

    pxz += dyyIn_0(0,0,-2)*c[-2+half_order];
    pyz += dyyIn_1(0,0,-2)*c[-2+half_order];
    pzz += dyyIn_2(0,0,-2)*c[-2+half_order];

    vxz += dyyIn_3(0,0,-2)*c[-2+half_order];
    vyz += dyyIn_4(0,0,-2)*c[-2+half_order];
    vzz += dyyIn_5(0,0,-2)*c[-2+half_order];

    pxx += dyyIn_0(-1,0,0)*c[-1+half_order];
    pyx += dyyIn_1(-1,0,0)*c[-1+half_order];
    pzx += dyyIn_2(-1,0,0)*c[-1+half_order];

    vxx += dyyIn_3(-1,0,0)*c[-1+half_order];
    vyx += dyyIn_4(-1,0,0)*c[-1+half_order];
    vzx += dyyIn_5(-1,0,0)*c[-1+half_order];

    pxy += dyyIn_0(0,-1,0)*c[-1+half_order];
    pyy += dyyIn_1(0,-1,0)*c[-1+half_order];
    pzy += dyyIn_2(0,-1,0)*c[-1+half_order];

    vxy += dyyIn_3(0,-1,0)*c[-1+half_order];
    vyy += dyyIn_4(0,-1,0)*c[-1+half_order];
    vzy += dyyIn_5(0,-1,0)*c[-1+half_order];

    pxz += dyyIn_0(0,0,-1)*c[-1+half_order];
    pyz += dyyIn_1(0,0,-1)*c[-1+half_order];
    pzz += dyyIn_2(0,0,-1)*c[-1+half_order];

    vxz += dyyIn_3(0,0,-1)*c[-1+half_order];
    vyz += dyyIn_4(0,0,-1)*c[-1+half_order];
    vzz += dyyIn_5(0,0,-1)*c[-1+half_order];

    pxx += dyyIn_0(0,0,0)*c[half_order];
    pyx += dyyIn_1(0,0,0)*c[half_order];
    pzx += dyyIn_2(0,0,0)*c[half_order];

    vxx += dyyIn_3(0,0,0)*c[half_order];
    vyx += dyyIn_4(0,0,0)*c[half_order];
    vzx += dyyIn_5(0,0,0)*c[half_order];

    pxy += dyyIn_0(0,0,0)*c[half_order];
    pyy += dyyIn_1(0,0,0)*c[half_order];
    pzy += dyyIn_2(0,0,0)*c[half_order];

    vxy += dyyIn_3(0,0,0)*c[half_order];
    vyy += dyyIn_4(0,0,0)*c[half_order];
    vzy += dyyIn_5(0,0,0)*c[half_order];

    pxz += dyyIn_0(0,0,0)*c[half_order];
    pyz += dyyIn_1(0,0,0)*c[half_order];
    pzz += dyyIn_2(0,0,0)*c[half_order];

    vxz += dyyIn_3(0,0,0)*c[half_order];
    vyz += dyyIn_4(0,0,0)*c[half_order];
    vzz += dyyIn_5(0,0,0)*c[half_order];

    pxx += dyyIn_0(1,0,0)*c[1+half_order];
    pyx += dyyIn_1(1,0,0)*c[1+half_order];
    pzx += dyyIn_2(1,0,0)*c[1+half_order];

    vxx += dyyIn_3(1,0,0)*c[1+half_order];
    vyx += dyyIn_4(1,0,0)*c[1+half_order];
    vzx += dyyIn_5(1,0,0)*c[1+half_order];

    pxy += dyyIn_0(0,1,0)*c[1+half_order];
    pyy += dyyIn_1(0,1,0)*c[1+half_order];
    pzy += dyyIn_2(0,1,0)*c[1+half_order];

    vxy += dyyIn_3(0,1,0)*c[1+half_order];
    vyy += dyyIn_4(0,1,0)*c[1+half_order];
    vzy += dyyIn_5(0,1,0)*c[1+half_order];

    pxz += dyyIn_0(0,0,1)*c[1+half_order];
    pyz += dyyIn_1(0,0,1)*c[1+half_order];
    pzz += dyyIn_2(0,0,1)*c[1+half_order];

    vxz += dyyIn_3(0,0,1)*c[1+half_order];
    vyz += dyyIn_4(0,0,1)*c[1+half_order];
    vzz += dyyIn_5(0,0,1)*c[1+half_order];

    pxx += dyyIn_0(2,0,0)*c[2+half_order];
    pyx += dyyIn_1(2,0,0)*c[2+half_order];
    pzx += dyyIn_2(2,0,0)*c[2+half_order];

    vxx += dyyIn_3(2,0,0)*c[2+half_order];
    vyx += dyyIn_4(2,0,0)*c[2+half_order];
    vzx += dyyIn_5(2,0,0)*c[2+half_order];

    pxy += dyyIn_0(0,2,0)*c[2+half_order];
    pyy += dyyIn_1(0,2,0)*c[2+half_order];
    pzy += dyyIn_2(0,2,0)*c[2+half_order];

    vxy += dyyIn_3(0,2,0)*c[2+half_order];
    vyy += dyyIn_4(0,2,0)*c[2+half_order];
    vzy += dyyIn_5(0,2,0)*c[2+half_order];

    pxz += dyyIn_0(0,0,2)*c[2+half_order];
    pyz += dyyIn_1(0,0,2)*c[2+half_order];
    pzz += dyyIn_2(0,0,2)*c[2+half_order];

    vxz += dyyIn_3(0,0,2)*c[2+half_order];
    vyz += dyyIn_4(0,0,2)*c[2+half_order];
    vzz += dyyIn_5(0,0,2)*c[2+half_order];

    pxx += dyyIn_0(3,0,0)*c[3+half_order];
    pyx += dyyIn_1(3,0,0)*c[3+half_order];
    pzx += dyyIn_2(3,0,0)*c[3+half_order];

    vxx += dyyIn_3(3,0,0)*c[3+half_order];
    vyx += dyyIn_4(3,0,0)*c[3+half_order];
    vzx += dyyIn_5(3,0,0)*c[3+half_order];

    pxy += dyyIn_0(0,3,0)*c[3+half_order];
    pyy += dyyIn_1(0,3,0)*c[3+half_order];
    pzy += dyyIn_2(0,3,0)*c[3+half_order];

    vxy += dyyIn_3(0,3,0)*c[3+half_order];
    vyy += dyyIn_4(0,3,0)*c[3+half_order];
    vzy += dyyIn_5(0,3,0)*c[3+half_order];

    pxz += dyyIn_0(0,0,3)*c[3+half_order];
    pyz += dyyIn_1(0,0,3)*c[3+half_order];
    pzz += dyyIn_2(0,0,3)*c[3+half_order];

    vxz += dyyIn_3(0,0,3)*c[3+half_order];
    vyz += dyyIn_4(0,0,3)*c[3+half_order];
    vzz += dyyIn_5(0,0,3)*c[3+half_order];

    pxx += dyyIn_0(4,0,0)*c[4+half_order];
    pyx += dyyIn_1(4,0,0)*c[4+half_order];
    pzx += dyyIn_2(4,0,0)*c[4+half_order];

    vxx += dyyIn_3(4,0,0)*c[4+half_order];
    vyx += dyyIn_4(4,0,0)*c[4+half_order];
    vzx += dyyIn_5(4,0,0)*c[4+half_order];

    pxy += dyyIn_0(0,4,0)*c[4+half_order];
    pyy += dyyIn_1(0,4,0)*c[4+half_order];
    pzy += dyyIn_2(0,4,0)*c[4+half_order];

    vxy += dyyIn_3(0,4,0)*c[4+half_order];
    vyy += dyyIn_4(0,4,0)*c[4+half_order];
    vzy += dyyIn_5(0,4,0)*c[4+half_order];

    pxz += dyyIn_0(0,0,4)*c[4+half_order];
    pyz += dyyIn_1(0,0,4)*c[4+half_order];
    pzz += dyyIn_2(0,0,4)*c[4+half_order];

    vxz += dyyIn_3(0,0,4)*c[4+half_order];
    vyz += dyyIn_4(0,0,4)*c[4+half_order];
    vzz += dyyIn_5(0,0,4)*c[4+half_order];

    pxx *= invdx;
    pyx *= invdx;
    pzx *= invdx;

    vxx *= invdx;
    vyx *= invdx;
    vzx *= invdx;

    pxy *= invdy;
    pyy *= invdy;
    pzy *= invdy;

    vxy *= invdy;
    vyy *= invdy;
    vzy *= invdy;

    pxz *= invdz;
    pyz *= invdz;
    pzz *= invdz;

    vxz *= invdz;
    vyz *= invdz;
    vzz *= invdz;

    float ytemp0 =(vxx/rho(0,0,0) - sigmax*px) * *dt;
    float ytemp3 =((pxx+pyx+pxz)*mu(0,0,0) - sigmax*vx)* *dt;

    float ytemp1 =(vyy/rho(0,0,0) - sigmay*py)* *dt;
    float ytemp4 =((pxy+pyy+pyz)*mu(0,0,0) - sigmay*vy)* *dt;

    float ytemp2 =(vzz/rho(0,0,0) - sigmaz*pz)* *dt;
    float ytemp5 =((pxz+pyz+pzz)*mu(0,0,0) - sigmaz*vz)* *dt;

    dyyOut_0(0,0,0) = yy_0(0,0,0) + ytemp0* *scale1;
    dyyOut_3(0,0,0) = yy_3(0,0,0) + ytemp3* *scale1;
    dyyOut_1(0,0,0) = yy_1(0,0,0) + ytemp1* *scale1;
    dyyOut_4(0,0,0) = yy_4(0,0,0) + ytemp4* *scale1;
    dyyOut_2(0,0,0) = yy_2(0,0,0) + ytemp2* *scale1;
    dyyOut_5(0,0,0) = yy_5(0,0,0) + ytemp5* *scale1;

    sum_0(0,0,0) += ytemp0 * *scale2;
    sum_3(0,0,0) += ytemp3 * *scale2;
    sum_1(0,0,0) += ytemp1 * *scale2;
    sum_4(0,0,0) += ytemp4 * *scale2;
    sum_2(0,0,0) += ytemp2 * *scale2;
    sum_5(0,0,0) += ytemp5 * *scale2;

}

//  ============================
//  Cuda kernel wrapper function
//  ============================
__global__ void ops_fd3d_pml_kernel2(const int arg0, 
const int arg1, 
const int arg2, 
int arg_idx0, int arg_idx1, int arg_idx2, 
const float arg4, 
const float arg5, 
const float arg6, 
float* __restrict arg7, int xstride_7, int ystride_7, int zstride_7, 
float* __restrict arg8, int xstride_8, int ystride_8, int zstride_8, 
float* __restrict arg9, int xstride_9, int ystride_9, int zstride_9, 
float* __restrict arg10, int xstride_10, int ystride_10, int zstride_10, 
float* __restrict arg11, int xstride_11, int ystride_11, int zstride_11, 
float* __restrict arg12, int xstride_12, int ystride_12, int zstride_12, 
float* __restrict arg13, int xstride_13, int ystride_13, int zstride_13, 
float* __restrict arg14, int xstride_14, int ystride_14, int zstride_14, 
float* __restrict arg15, int xstride_15, int ystride_15, int zstride_15, 
float* __restrict arg16, int xstride_16, int ystride_16, int zstride_16, 
float* __restrict arg17, int xstride_17, int ystride_17, int zstride_17, 
float* __restrict arg18, int xstride_18, int ystride_18, int zstride_18, 
float* __restrict arg19, int xstride_19, int ystride_19, int zstride_19, 
float* __restrict arg20, int xstride_20, int ystride_20, int zstride_20, 
float* __restrict arg21, int xstride_21, int ystride_21, int zstride_21, 
float* __restrict arg22, int xstride_22, int ystride_22, int zstride_22, 
float* __restrict arg23, int xstride_23, int ystride_23, int zstride_23, 
float* __restrict arg24, int xstride_24, int ystride_24, int zstride_24, 
float* __restrict arg25, int xstride_25, int ystride_25, int zstride_25, 
float* __restrict arg26, int xstride_26, int ystride_26, int zstride_26, 
float* __restrict arg27, int xstride_27, int ystride_27, int zstride_27, 
float* __restrict arg28, int xstride_28, int ystride_28, int zstride_28, 
float* __restrict arg29, int xstride_29, int ystride_29, int zstride_29, 
float* __restrict arg30, int xstride_30, int ystride_30, int zstride_30, 
float* __restrict arg31, int xstride_31, int ystride_31, int zstride_31, 
float* __restrict arg32, int xstride_32, int ystride_32, int zstride_32, 
int size0, int size1, int size2) {

    int idx_z = blockDim.z * blockIdx.z + threadIdx.z;
    int idx_y = blockDim.y * blockIdx.y + threadIdx.y;
    int idx_x = blockDim.x * blockIdx.x + threadIdx.x;

    int arg_idx[3];
    arg_idx[0] = arg_idx0+idx_x;
    arg_idx[1] = arg_idx1+idx_y;
    arg_idx[2] = arg_idx2+idx_z;

    arg7 += idx_x * xstride_7*1 + idx_y * ystride_7*1 * dims_fd3d_pml_kernel2[7][0] + idx_z * zstride_7*1 * dims_fd3d_pml_kernel2[7][0] * dims_fd3d_pml_kernel2[7][1];
    arg8 += idx_x * xstride_8*1 + idx_y * ystride_8*1 * dims_fd3d_pml_kernel2[8][0] + idx_z * zstride_8*1 * dims_fd3d_pml_kernel2[8][0] * dims_fd3d_pml_kernel2[8][1];
    arg9 += idx_x * xstride_9*1 + idx_y * ystride_9*1 * dims_fd3d_pml_kernel2[9][0] + idx_z * zstride_9*1 * dims_fd3d_pml_kernel2[9][0] * dims_fd3d_pml_kernel2[9][1];
    arg10 += idx_x * xstride_10*1 + idx_y * ystride_10*1 * dims_fd3d_pml_kernel2[10][0] + idx_z * zstride_10*1 * dims_fd3d_pml_kernel2[10][0] * dims_fd3d_pml_kernel2[10][1];
    arg11 += idx_x * xstride_11*1 + idx_y * ystride_11*1 * dims_fd3d_pml_kernel2[11][0] + idx_z * zstride_11*1 * dims_fd3d_pml_kernel2[11][0] * dims_fd3d_pml_kernel2[11][1];
    arg12 += idx_x * xstride_12*1 + idx_y * ystride_12*1 * dims_fd3d_pml_kernel2[12][0] + idx_z * zstride_12*1 * dims_fd3d_pml_kernel2[12][0] * dims_fd3d_pml_kernel2[12][1];
    arg13 += idx_x * xstride_13*1 + idx_y * ystride_13*1 * dims_fd3d_pml_kernel2[13][0] + idx_z * zstride_13*1 * dims_fd3d_pml_kernel2[13][0] * dims_fd3d_pml_kernel2[13][1];
    arg14 += idx_x * xstride_14*1 + idx_y * ystride_14*1 * dims_fd3d_pml_kernel2[14][0] + idx_z * zstride_14*1 * dims_fd3d_pml_kernel2[14][0] * dims_fd3d_pml_kernel2[14][1];
    arg15 += idx_x * xstride_15*1 + idx_y * ystride_15*1 * dims_fd3d_pml_kernel2[15][0] + idx_z * zstride_15*1 * dims_fd3d_pml_kernel2[15][0] * dims_fd3d_pml_kernel2[15][1];
    arg16 += idx_x * xstride_16*1 + idx_y * ystride_16*1 * dims_fd3d_pml_kernel2[16][0] + idx_z * zstride_16*1 * dims_fd3d_pml_kernel2[16][0] * dims_fd3d_pml_kernel2[16][1];
    arg17 += idx_x * xstride_17*1 + idx_y * ystride_17*1 * dims_fd3d_pml_kernel2[17][0] + idx_z * zstride_17*1 * dims_fd3d_pml_kernel2[17][0] * dims_fd3d_pml_kernel2[17][1];
    arg18 += idx_x * xstride_18*1 + idx_y * ystride_18*1 * dims_fd3d_pml_kernel2[18][0] + idx_z * zstride_18*1 * dims_fd3d_pml_kernel2[18][0] * dims_fd3d_pml_kernel2[18][1];
    arg19 += idx_x * xstride_19*1 + idx_y * ystride_19*1 * dims_fd3d_pml_kernel2[19][0] + idx_z * zstride_19*1 * dims_fd3d_pml_kernel2[19][0] * dims_fd3d_pml_kernel2[19][1];
    arg20 += idx_x * xstride_20*1 + idx_y * ystride_20*1 * dims_fd3d_pml_kernel2[20][0] + idx_z * zstride_20*1 * dims_fd3d_pml_kernel2[20][0] * dims_fd3d_pml_kernel2[20][1];
    arg21 += idx_x * xstride_21*1 + idx_y * ystride_21*1 * dims_fd3d_pml_kernel2[21][0] + idx_z * zstride_21*1 * dims_fd3d_pml_kernel2[21][0] * dims_fd3d_pml_kernel2[21][1];
    arg22 += idx_x * xstride_22*1 + idx_y * ystride_22*1 * dims_fd3d_pml_kernel2[22][0] + idx_z * zstride_22*1 * dims_fd3d_pml_kernel2[22][0] * dims_fd3d_pml_kernel2[22][1];
    arg23 += idx_x * xstride_23*1 + idx_y * ystride_23*1 * dims_fd3d_pml_kernel2[23][0] + idx_z * zstride_23*1 * dims_fd3d_pml_kernel2[23][0] * dims_fd3d_pml_kernel2[23][1];
    arg24 += idx_x * xstride_24*1 + idx_y * ystride_24*1 * dims_fd3d_pml_kernel2[24][0] + idx_z * zstride_24*1 * dims_fd3d_pml_kernel2[24][0] * dims_fd3d_pml_kernel2[24][1];
    arg25 += idx_x * xstride_25*1 + idx_y * ystride_25*1 * dims_fd3d_pml_kernel2[25][0] + idx_z * zstride_25*1 * dims_fd3d_pml_kernel2[25][0] * dims_fd3d_pml_kernel2[25][1];
    arg26 += idx_x * xstride_26*1 + idx_y * ystride_26*1 * dims_fd3d_pml_kernel2[26][0] + idx_z * zstride_26*1 * dims_fd3d_pml_kernel2[26][0] * dims_fd3d_pml_kernel2[26][1];
    arg27 += idx_x * xstride_27*1 + idx_y * ystride_27*1 * dims_fd3d_pml_kernel2[27][0] + idx_z * zstride_27*1 * dims_fd3d_pml_kernel2[27][0] * dims_fd3d_pml_kernel2[27][1];
    arg28 += idx_x * xstride_28*1 + idx_y * ystride_28*1 * dims_fd3d_pml_kernel2[28][0] + idx_z * zstride_28*1 * dims_fd3d_pml_kernel2[28][0] * dims_fd3d_pml_kernel2[28][1];
    arg29 += idx_x * xstride_29*1 + idx_y * ystride_29*1 * dims_fd3d_pml_kernel2[29][0] + idx_z * zstride_29*1 * dims_fd3d_pml_kernel2[29][0] * dims_fd3d_pml_kernel2[29][1];
    arg30 += idx_x * xstride_30*1 + idx_y * ystride_30*1 * dims_fd3d_pml_kernel2[30][0] + idx_z * zstride_30*1 * dims_fd3d_pml_kernel2[30][0] * dims_fd3d_pml_kernel2[30][1];
    arg31 += idx_x * xstride_31*1 + idx_y * ystride_31*1 * dims_fd3d_pml_kernel2[31][0] + idx_z * zstride_31*1 * dims_fd3d_pml_kernel2[31][0] * dims_fd3d_pml_kernel2[31][1];
    arg32 += idx_x * xstride_32*1 + idx_y * ystride_32*1 * dims_fd3d_pml_kernel2[32][0] + idx_z * zstride_32*1 * dims_fd3d_pml_kernel2[32][0] * dims_fd3d_pml_kernel2[32][1];

    if(idx_x < size0 && idx_y < size1 && idx_z < size2) {

        const ACC<float> argp7(dims_fd3d_pml_kernel2[7][0], dims_fd3d_pml_kernel2[7][1], arg7);
        const ACC<float> argp8(dims_fd3d_pml_kernel2[8][0], dims_fd3d_pml_kernel2[8][1], arg8);
        const ACC<float> argp9(dims_fd3d_pml_kernel2[9][0], dims_fd3d_pml_kernel2[9][1], arg9);
        const ACC<float> argp10(dims_fd3d_pml_kernel2[10][0], dims_fd3d_pml_kernel2[10][1], arg10);
        const ACC<float> argp11(dims_fd3d_pml_kernel2[11][0], dims_fd3d_pml_kernel2[11][1], arg11);
        const ACC<float> argp12(dims_fd3d_pml_kernel2[12][0], dims_fd3d_pml_kernel2[12][1], arg12);
        const ACC<float> argp13(dims_fd3d_pml_kernel2[13][0], dims_fd3d_pml_kernel2[13][1], arg13);
        const ACC<float> argp14(dims_fd3d_pml_kernel2[14][0], dims_fd3d_pml_kernel2[14][1], arg14);
        const ACC<float> argp15(dims_fd3d_pml_kernel2[15][0], dims_fd3d_pml_kernel2[15][1], arg15);
        const ACC<float> argp16(dims_fd3d_pml_kernel2[16][0], dims_fd3d_pml_kernel2[16][1], arg16);
        const ACC<float> argp17(dims_fd3d_pml_kernel2[17][0], dims_fd3d_pml_kernel2[17][1], arg17);
        const ACC<float> argp18(dims_fd3d_pml_kernel2[18][0], dims_fd3d_pml_kernel2[18][1], arg18);
        const ACC<float> argp19(dims_fd3d_pml_kernel2[19][0], dims_fd3d_pml_kernel2[19][1], arg19);
        const ACC<float> argp20(dims_fd3d_pml_kernel2[20][0], dims_fd3d_pml_kernel2[20][1], arg20);
        ACC<float> argp21(dims_fd3d_pml_kernel2[21][0], dims_fd3d_pml_kernel2[21][1], arg21);
        ACC<float> argp22(dims_fd3d_pml_kernel2[22][0], dims_fd3d_pml_kernel2[22][1], arg22);
        ACC<float> argp23(dims_fd3d_pml_kernel2[23][0], dims_fd3d_pml_kernel2[23][1], arg23);
        ACC<float> argp24(dims_fd3d_pml_kernel2[24][0], dims_fd3d_pml_kernel2[24][1], arg24);
        ACC<float> argp25(dims_fd3d_pml_kernel2[25][0], dims_fd3d_pml_kernel2[25][1], arg25);
        ACC<float> argp26(dims_fd3d_pml_kernel2[26][0], dims_fd3d_pml_kernel2[26][1], arg26);
        ACC<float> argp27(dims_fd3d_pml_kernel2[27][0], dims_fd3d_pml_kernel2[27][1], arg27);
        ACC<float> argp28(dims_fd3d_pml_kernel2[28][0], dims_fd3d_pml_kernel2[28][1], arg28);
        ACC<float> argp29(dims_fd3d_pml_kernel2[29][0], dims_fd3d_pml_kernel2[29][1], arg29);
        ACC<float> argp30(dims_fd3d_pml_kernel2[30][0], dims_fd3d_pml_kernel2[30][1], arg30);
        ACC<float> argp31(dims_fd3d_pml_kernel2[31][0], dims_fd3d_pml_kernel2[31][1], arg31);
        ACC<float> argp32(dims_fd3d_pml_kernel2[32][0], dims_fd3d_pml_kernel2[32][1], arg32);

        fd3d_pml_kernel2_gpu(&arg0, &arg1, &arg2, arg_idx, &arg4, &arg5, &arg6, argp7, argp8, argp9, argp10, argp11, argp12, argp13, argp14, argp15, argp16, argp17, argp18, argp19, argp20, argp21, argp22, argp23, argp24, argp25, argp26, argp27, argp28, argp29, argp30, argp31, argp32);

    }// End of cuda index in_range check

}// End of cuda kernel wrapper function

//  ==================
//  Host stub function
//  ==================
#ifndef OPS_LAZY
void ops_par_loop_fd3d_pml_kernel2(
    const char * name,
    ops_block block,
    int dim,
    int *range,
    ops_arg arg0,
    ops_arg arg1,
    ops_arg arg2,
    ops_arg arg3,
    ops_arg arg4,
    ops_arg arg5,
    ops_arg arg6,
    ops_arg arg7,
    ops_arg arg8,
    ops_arg arg9,
    ops_arg arg10,
    ops_arg arg11,
    ops_arg arg12,
    ops_arg arg13,
    ops_arg arg14,
    ops_arg arg15,
    ops_arg arg16,
    ops_arg arg17,
    ops_arg arg18,
    ops_arg arg19,
    ops_arg arg20,
    ops_arg arg21,
    ops_arg arg22,
    ops_arg arg23,
    ops_arg arg24,
    ops_arg arg25,
    ops_arg arg26,
    ops_arg arg27,
    ops_arg arg28,
    ops_arg arg29,
    ops_arg arg30,
    ops_arg arg31,
    ops_arg arg32
)
{ 
#else
void ops_par_loop_fd3d_pml_kernel2_execute(ops_kernel_descriptor *desc)
{
    ops_block block = desc->block;
    int dim = desc->dim;
    int *range = desc->range;
    ops_arg arg0 = desc->args[0];
    ops_arg arg1 = desc->args[1];
    ops_arg arg2 = desc->args[2];
    ops_arg arg3 = desc->args[3];
    ops_arg arg4 = desc->args[4];
    ops_arg arg5 = desc->args[5];
    ops_arg arg6 = desc->args[6];
    ops_arg arg7 = desc->args[7];
    ops_arg arg8 = desc->args[8];
    ops_arg arg9 = desc->args[9];
    ops_arg arg10 = desc->args[10];
    ops_arg arg11 = desc->args[11];
    ops_arg arg12 = desc->args[12];
    ops_arg arg13 = desc->args[13];
    ops_arg arg14 = desc->args[14];
    ops_arg arg15 = desc->args[15];
    ops_arg arg16 = desc->args[16];
    ops_arg arg17 = desc->args[17];
    ops_arg arg18 = desc->args[18];
    ops_arg arg19 = desc->args[19];
    ops_arg arg20 = desc->args[20];
    ops_arg arg21 = desc->args[21];
    ops_arg arg22 = desc->args[22];
    ops_arg arg23 = desc->args[23];
    ops_arg arg24 = desc->args[24];
    ops_arg arg25 = desc->args[25];
    ops_arg arg26 = desc->args[26];
    ops_arg arg27 = desc->args[27];
    ops_arg arg28 = desc->args[28];
    ops_arg arg29 = desc->args[29];
    ops_arg arg30 = desc->args[30];
    ops_arg arg31 = desc->args[31];
    ops_arg arg32 = desc->args[32];
#endif

//  ======
//  Timing
//  ======
    double __t1, __t2, __c1, __c2;

    ops_arg args[33];

    args[0] = arg0;
    args[1] = arg1;
    args[2] = arg2;
    args[3] = arg3;
    args[4] = arg4;
    args[5] = arg5;
    args[6] = arg6;
    args[7] = arg7;
    args[8] = arg8;
    args[9] = arg9;
    args[10] = arg10;
    args[11] = arg11;
    args[12] = arg12;
    args[13] = arg13;
    args[14] = arg14;
    args[15] = arg15;
    args[16] = arg16;
    args[17] = arg17;
    args[18] = arg18;
    args[19] = arg19;
    args[20] = arg20;
    args[21] = arg21;
    args[22] = arg22;
    args[23] = arg23;
    args[24] = arg24;
    args[25] = arg25;
    args[26] = arg26;
    args[27] = arg27;
    args[28] = arg28;
    args[29] = arg29;
    args[30] = arg30;
    args[31] = arg31;
    args[32] = arg32;

#if defined(CHECKPOINTING) && !defined(OPS_LAZY)
    if (!ops_checkpointing_before(args, 33, range, 4)) return;
#endif

    if (block->instance->OPS_diags > 1)
    {
        ops_timing_realloc(block->instance, 4, "fd3d_pml_kernel2");
        block->instance->OPS_kernels[4].count++;
        ops_timers_core(&__c1, &__t1);
    }

//  =================================================
//  compute locally allocated range for the sub-block
//  =================================================
    int start_indx[3];
    int end_indx[3];
    int arg_idx[3];

#if defined(OPS_LAZY) || !defined(OPS_MPI)
    for (int n = 0; n < 3; n++) {
        start_indx[n] = range[2*n];
        end_indx[n]   = range[2*n+1];
    }
#else
    if (compute_ranges(args, 33, block, range, start_indx, end_indx, arg_idx) < 0) return;
#endif

#if defined(OPS_MPI)
#if defined(OPS_LAZY)
    sub_block_list sb = OPS_sub_block_list[block->index];
    arg_idx[0] = sb->decomp_disp[0]+start_indx[0];
    arg_idx[1] = sb->decomp_disp[1]+start_indx[1];
    arg_idx[2] = sb->decomp_disp[2]+start_indx[2];
#endif  // OPS_LAZY
#else   // OPS_MPI
    arg_idx[0] = start_indx[0];
    arg_idx[1] = start_indx[1];
    arg_idx[2] = start_indx[2];
#endif  // OPS_MPI

    int xdim7 = args[7].dat->size[0];
    int ydim7 = args[7].dat->size[1];
    int xdim8 = args[8].dat->size[0];
    int ydim8 = args[8].dat->size[1];
    int xdim9 = args[9].dat->size[0];
    int ydim9 = args[9].dat->size[1];
    int xdim10 = args[10].dat->size[0];
    int ydim10 = args[10].dat->size[1];
    int xdim11 = args[11].dat->size[0];
    int ydim11 = args[11].dat->size[1];
    int xdim12 = args[12].dat->size[0];
    int ydim12 = args[12].dat->size[1];
    int xdim13 = args[13].dat->size[0];
    int ydim13 = args[13].dat->size[1];
    int xdim14 = args[14].dat->size[0];
    int ydim14 = args[14].dat->size[1];
    int xdim15 = args[15].dat->size[0];
    int ydim15 = args[15].dat->size[1];
    int xdim16 = args[16].dat->size[0];
    int ydim16 = args[16].dat->size[1];
    int xdim17 = args[17].dat->size[0];
    int ydim17 = args[17].dat->size[1];
    int xdim18 = args[18].dat->size[0];
    int ydim18 = args[18].dat->size[1];
    int xdim19 = args[19].dat->size[0];
    int ydim19 = args[19].dat->size[1];
    int xdim20 = args[20].dat->size[0];
    int ydim20 = args[20].dat->size[1];
    int xdim21 = args[21].dat->size[0];
    int ydim21 = args[21].dat->size[1];
    int xdim22 = args[22].dat->size[0];
    int ydim22 = args[22].dat->size[1];
    int xdim23 = args[23].dat->size[0];
    int ydim23 = args[23].dat->size[1];
    int xdim24 = args[24].dat->size[0];
    int ydim24 = args[24].dat->size[1];
    int xdim25 = args[25].dat->size[0];
    int ydim25 = args[25].dat->size[1];
    int xdim26 = args[26].dat->size[0];
    int ydim26 = args[26].dat->size[1];
    int xdim27 = args[27].dat->size[0];
    int ydim27 = args[27].dat->size[1];
    int xdim28 = args[28].dat->size[0];
    int ydim28 = args[28].dat->size[1];
    int xdim29 = args[29].dat->size[0];
    int ydim29 = args[29].dat->size[1];
    int xdim30 = args[30].dat->size[0];
    int ydim30 = args[30].dat->size[1];
    int xdim31 = args[31].dat->size[0];
    int ydim31 = args[31].dat->size[1];
    int xdim32 = args[32].dat->size[0];
    int ydim32 = args[32].dat->size[1];

    if (xdim7 != dims_fd3d_pml_kernel2_h[7][0] || ydim7 != dims_fd3d_pml_kernel2_h[7][1] || xdim8 != dims_fd3d_pml_kernel2_h[8][0] || ydim8 != dims_fd3d_pml_kernel2_h[8][1] || xdim9 != dims_fd3d_pml_kernel2_h[9][0] || ydim9 != dims_fd3d_pml_kernel2_h[9][1] || xdim10 != dims_fd3d_pml_kernel2_h[10][0] || ydim10 != dims_fd3d_pml_kernel2_h[10][1] || xdim11 != dims_fd3d_pml_kernel2_h[11][0] || ydim11 != dims_fd3d_pml_kernel2_h[11][1] || xdim12 != dims_fd3d_pml_kernel2_h[12][0] || ydim12 != dims_fd3d_pml_kernel2_h[12][1] || xdim13 != dims_fd3d_pml_kernel2_h[13][0] || ydim13 != dims_fd3d_pml_kernel2_h[13][1] || xdim14 != dims_fd3d_pml_kernel2_h[14][0] || ydim14 != dims_fd3d_pml_kernel2_h[14][1] || xdim15 != dims_fd3d_pml_kernel2_h[15][0] || ydim15 != dims_fd3d_pml_kernel2_h[15][1] || xdim16 != dims_fd3d_pml_kernel2_h[16][0] || ydim16 != dims_fd3d_pml_kernel2_h[16][1] || xdim17 != dims_fd3d_pml_kernel2_h[17][0] || ydim17 != dims_fd3d_pml_kernel2_h[17][1] || xdim18 != dims_fd3d_pml_kernel2_h[18][0] || ydim18 != dims_fd3d_pml_kernel2_h[18][1] || xdim19 != dims_fd3d_pml_kernel2_h[19][0] || ydim19 != dims_fd3d_pml_kernel2_h[19][1] || xdim20 != dims_fd3d_pml_kernel2_h[20][0] || ydim20 != dims_fd3d_pml_kernel2_h[20][1] || xdim21 != dims_fd3d_pml_kernel2_h[21][0] || ydim21 != dims_fd3d_pml_kernel2_h[21][1] || xdim22 != dims_fd3d_pml_kernel2_h[22][0] || ydim22 != dims_fd3d_pml_kernel2_h[22][1] || xdim23 != dims_fd3d_pml_kernel2_h[23][0] || ydim23 != dims_fd3d_pml_kernel2_h[23][1] || xdim24 != dims_fd3d_pml_kernel2_h[24][0] || ydim24 != dims_fd3d_pml_kernel2_h[24][1] || xdim25 != dims_fd3d_pml_kernel2_h[25][0] || ydim25 != dims_fd3d_pml_kernel2_h[25][1] || xdim26 != dims_fd3d_pml_kernel2_h[26][0] || ydim26 != dims_fd3d_pml_kernel2_h[26][1] || xdim27 != dims_fd3d_pml_kernel2_h[27][0] || ydim27 != dims_fd3d_pml_kernel2_h[27][1] || xdim28 != dims_fd3d_pml_kernel2_h[28][0] || ydim28 != dims_fd3d_pml_kernel2_h[28][1] || xdim29 != dims_fd3d_pml_kernel2_h[29][0] || ydim29 != dims_fd3d_pml_kernel2_h[29][1] || xdim30 != dims_fd3d_pml_kernel2_h[30][0] || ydim30 != dims_fd3d_pml_kernel2_h[30][1] || xdim31 != dims_fd3d_pml_kernel2_h[31][0] || ydim31 != dims_fd3d_pml_kernel2_h[31][1] || xdim32 != dims_fd3d_pml_kernel2_h[32][0] || ydim32 != dims_fd3d_pml_kernel2_h[32][1]) {
        dims_fd3d_pml_kernel2_h[7][0] = xdim7;
        dims_fd3d_pml_kernel2_h[7][1] = ydim7;
        dims_fd3d_pml_kernel2_h[8][0] = xdim8;
        dims_fd3d_pml_kernel2_h[8][1] = ydim8;
        dims_fd3d_pml_kernel2_h[9][0] = xdim9;
        dims_fd3d_pml_kernel2_h[9][1] = ydim9;
        dims_fd3d_pml_kernel2_h[10][0] = xdim10;
        dims_fd3d_pml_kernel2_h[10][1] = ydim10;
        dims_fd3d_pml_kernel2_h[11][0] = xdim11;
        dims_fd3d_pml_kernel2_h[11][1] = ydim11;
        dims_fd3d_pml_kernel2_h[12][0] = xdim12;
        dims_fd3d_pml_kernel2_h[12][1] = ydim12;
        dims_fd3d_pml_kernel2_h[13][0] = xdim13;
        dims_fd3d_pml_kernel2_h[13][1] = ydim13;
        dims_fd3d_pml_kernel2_h[14][0] = xdim14;
        dims_fd3d_pml_kernel2_h[14][1] = ydim14;
        dims_fd3d_pml_kernel2_h[15][0] = xdim15;
        dims_fd3d_pml_kernel2_h[15][1] = ydim15;
        dims_fd3d_pml_kernel2_h[16][0] = xdim16;
        dims_fd3d_pml_kernel2_h[16][1] = ydim16;
        dims_fd3d_pml_kernel2_h[17][0] = xdim17;
        dims_fd3d_pml_kernel2_h[17][1] = ydim17;
        dims_fd3d_pml_kernel2_h[18][0] = xdim18;
        dims_fd3d_pml_kernel2_h[18][1] = ydim18;
        dims_fd3d_pml_kernel2_h[19][0] = xdim19;
        dims_fd3d_pml_kernel2_h[19][1] = ydim19;
        dims_fd3d_pml_kernel2_h[20][0] = xdim20;
        dims_fd3d_pml_kernel2_h[20][1] = ydim20;
        dims_fd3d_pml_kernel2_h[21][0] = xdim21;
        dims_fd3d_pml_kernel2_h[21][1] = ydim21;
        dims_fd3d_pml_kernel2_h[22][0] = xdim22;
        dims_fd3d_pml_kernel2_h[22][1] = ydim22;
        dims_fd3d_pml_kernel2_h[23][0] = xdim23;
        dims_fd3d_pml_kernel2_h[23][1] = ydim23;
        dims_fd3d_pml_kernel2_h[24][0] = xdim24;
        dims_fd3d_pml_kernel2_h[24][1] = ydim24;
        dims_fd3d_pml_kernel2_h[25][0] = xdim25;
        dims_fd3d_pml_kernel2_h[25][1] = ydim25;
        dims_fd3d_pml_kernel2_h[26][0] = xdim26;
        dims_fd3d_pml_kernel2_h[26][1] = ydim26;
        dims_fd3d_pml_kernel2_h[27][0] = xdim27;
        dims_fd3d_pml_kernel2_h[27][1] = ydim27;
        dims_fd3d_pml_kernel2_h[28][0] = xdim28;
        dims_fd3d_pml_kernel2_h[28][1] = ydim28;
        dims_fd3d_pml_kernel2_h[29][0] = xdim29;
        dims_fd3d_pml_kernel2_h[29][1] = ydim29;
        dims_fd3d_pml_kernel2_h[30][0] = xdim30;
        dims_fd3d_pml_kernel2_h[30][1] = ydim30;
        dims_fd3d_pml_kernel2_h[31][0] = xdim31;
        dims_fd3d_pml_kernel2_h[31][1] = ydim31;
        dims_fd3d_pml_kernel2_h[32][0] = xdim32;
        dims_fd3d_pml_kernel2_h[32][1] = ydim32;

        cutilSafeCall(block->instance->ostream(), hipMemcpyToSymbol(HIP_SYMBOL( dims_fd3d_pml_kernel2), dims_fd3d_pml_kernel2_h, sizeof(dims_fd3d_pml_kernel2)));
    }

    int x_size = MAX(0,end_indx[0]-start_indx[0]);
    int y_size = MAX(0,end_indx[1]-start_indx[1]);
    int z_size = MAX(0,end_indx[2]-start_indx[2]);

    dim3 grid( (x_size-1)/block->instance->OPS_block_size_x + 1, (y_size-1)/block->instance->OPS_block_size_y + 1, (z_size-1)/block->instance->OPS_block_size_z + 1);

    dim3 tblock(block->instance->OPS_block_size_x,block->instance->OPS_block_size_y,block->instance->OPS_block_size_z);

    long long int dat7 = (block->instance->OPS_soa ? args[7].dat->type_size : args[7].dat->elem_size);
    long long int dat8 = (block->instance->OPS_soa ? args[8].dat->type_size : args[8].dat->elem_size);
    long long int dat9 = (block->instance->OPS_soa ? args[9].dat->type_size : args[9].dat->elem_size);
    long long int dat10 = (block->instance->OPS_soa ? args[10].dat->type_size : args[10].dat->elem_size);
    long long int dat11 = (block->instance->OPS_soa ? args[11].dat->type_size : args[11].dat->elem_size);
    long long int dat12 = (block->instance->OPS_soa ? args[12].dat->type_size : args[12].dat->elem_size);
    long long int dat13 = (block->instance->OPS_soa ? args[13].dat->type_size : args[13].dat->elem_size);
    long long int dat14 = (block->instance->OPS_soa ? args[14].dat->type_size : args[14].dat->elem_size);
    long long int dat15 = (block->instance->OPS_soa ? args[15].dat->type_size : args[15].dat->elem_size);
    long long int dat16 = (block->instance->OPS_soa ? args[16].dat->type_size : args[16].dat->elem_size);
    long long int dat17 = (block->instance->OPS_soa ? args[17].dat->type_size : args[17].dat->elem_size);
    long long int dat18 = (block->instance->OPS_soa ? args[18].dat->type_size : args[18].dat->elem_size);
    long long int dat19 = (block->instance->OPS_soa ? args[19].dat->type_size : args[19].dat->elem_size);
    long long int dat20 = (block->instance->OPS_soa ? args[20].dat->type_size : args[20].dat->elem_size);
    long long int dat21 = (block->instance->OPS_soa ? args[21].dat->type_size : args[21].dat->elem_size);
    long long int dat22 = (block->instance->OPS_soa ? args[22].dat->type_size : args[22].dat->elem_size);
    long long int dat23 = (block->instance->OPS_soa ? args[23].dat->type_size : args[23].dat->elem_size);
    long long int dat24 = (block->instance->OPS_soa ? args[24].dat->type_size : args[24].dat->elem_size);
    long long int dat25 = (block->instance->OPS_soa ? args[25].dat->type_size : args[25].dat->elem_size);
    long long int dat26 = (block->instance->OPS_soa ? args[26].dat->type_size : args[26].dat->elem_size);
    long long int dat27 = (block->instance->OPS_soa ? args[27].dat->type_size : args[27].dat->elem_size);
    long long int dat28 = (block->instance->OPS_soa ? args[28].dat->type_size : args[28].dat->elem_size);
    long long int dat29 = (block->instance->OPS_soa ? args[29].dat->type_size : args[29].dat->elem_size);
    long long int dat30 = (block->instance->OPS_soa ? args[30].dat->type_size : args[30].dat->elem_size);
    long long int dat31 = (block->instance->OPS_soa ? args[31].dat->type_size : args[31].dat->elem_size);
    long long int dat32 = (block->instance->OPS_soa ? args[32].dat->type_size : args[32].dat->elem_size);

    char *p_a[33];

//  =======================
//  set up initial pointers
//  =======================
    long long int base7 = args[7].dat->base_offset + dat7 * 1 * (start_indx[0] * args[7].stencil->stride[0]);
    base7 = base7 + dat7 * 
                     args[7].dat->size[0] * 
                     (start_indx[1] * args[7].stencil->stride[1]);
    base7 = base7 + dat7 * 
                     args[7].dat->size[0] * 
                     args[7].dat->size[1] * 
                     (start_indx[2] * args[7].stencil->stride[2]);
    p_a[7] = (char *)args[7].data_d + base7;

    long long int base8 = args[8].dat->base_offset + dat8 * 1 * (start_indx[0] * args[8].stencil->stride[0]);
    base8 = base8 + dat8 * 
                     args[8].dat->size[0] * 
                     (start_indx[1] * args[8].stencil->stride[1]);
    base8 = base8 + dat8 * 
                     args[8].dat->size[0] * 
                     args[8].dat->size[1] * 
                     (start_indx[2] * args[8].stencil->stride[2]);
    p_a[8] = (char *)args[8].data_d + base8;

    long long int base9 = args[9].dat->base_offset + dat9 * 1 * (start_indx[0] * args[9].stencil->stride[0]);
    base9 = base9 + dat9 * 
                     args[9].dat->size[0] * 
                     (start_indx[1] * args[9].stencil->stride[1]);
    base9 = base9 + dat9 * 
                     args[9].dat->size[0] * 
                     args[9].dat->size[1] * 
                     (start_indx[2] * args[9].stencil->stride[2]);
    p_a[9] = (char *)args[9].data_d + base9;

    long long int base10 = args[10].dat->base_offset + dat10 * 1 * (start_indx[0] * args[10].stencil->stride[0]);
    base10 = base10 + dat10 * 
                     args[10].dat->size[0] * 
                     (start_indx[1] * args[10].stencil->stride[1]);
    base10 = base10 + dat10 * 
                     args[10].dat->size[0] * 
                     args[10].dat->size[1] * 
                     (start_indx[2] * args[10].stencil->stride[2]);
    p_a[10] = (char *)args[10].data_d + base10;

    long long int base11 = args[11].dat->base_offset + dat11 * 1 * (start_indx[0] * args[11].stencil->stride[0]);
    base11 = base11 + dat11 * 
                     args[11].dat->size[0] * 
                     (start_indx[1] * args[11].stencil->stride[1]);
    base11 = base11 + dat11 * 
                     args[11].dat->size[0] * 
                     args[11].dat->size[1] * 
                     (start_indx[2] * args[11].stencil->stride[2]);
    p_a[11] = (char *)args[11].data_d + base11;

    long long int base12 = args[12].dat->base_offset + dat12 * 1 * (start_indx[0] * args[12].stencil->stride[0]);
    base12 = base12 + dat12 * 
                     args[12].dat->size[0] * 
                     (start_indx[1] * args[12].stencil->stride[1]);
    base12 = base12 + dat12 * 
                     args[12].dat->size[0] * 
                     args[12].dat->size[1] * 
                     (start_indx[2] * args[12].stencil->stride[2]);
    p_a[12] = (char *)args[12].data_d + base12;

    long long int base13 = args[13].dat->base_offset + dat13 * 1 * (start_indx[0] * args[13].stencil->stride[0]);
    base13 = base13 + dat13 * 
                     args[13].dat->size[0] * 
                     (start_indx[1] * args[13].stencil->stride[1]);
    base13 = base13 + dat13 * 
                     args[13].dat->size[0] * 
                     args[13].dat->size[1] * 
                     (start_indx[2] * args[13].stencil->stride[2]);
    p_a[13] = (char *)args[13].data_d + base13;

    long long int base14 = args[14].dat->base_offset + dat14 * 1 * (start_indx[0] * args[14].stencil->stride[0]);
    base14 = base14 + dat14 * 
                     args[14].dat->size[0] * 
                     (start_indx[1] * args[14].stencil->stride[1]);
    base14 = base14 + dat14 * 
                     args[14].dat->size[0] * 
                     args[14].dat->size[1] * 
                     (start_indx[2] * args[14].stencil->stride[2]);
    p_a[14] = (char *)args[14].data_d + base14;

    long long int base15 = args[15].dat->base_offset + dat15 * 1 * (start_indx[0] * args[15].stencil->stride[0]);
    base15 = base15 + dat15 * 
                     args[15].dat->size[0] * 
                     (start_indx[1] * args[15].stencil->stride[1]);
    base15 = base15 + dat15 * 
                     args[15].dat->size[0] * 
                     args[15].dat->size[1] * 
                     (start_indx[2] * args[15].stencil->stride[2]);
    p_a[15] = (char *)args[15].data_d + base15;

    long long int base16 = args[16].dat->base_offset + dat16 * 1 * (start_indx[0] * args[16].stencil->stride[0]);
    base16 = base16 + dat16 * 
                     args[16].dat->size[0] * 
                     (start_indx[1] * args[16].stencil->stride[1]);
    base16 = base16 + dat16 * 
                     args[16].dat->size[0] * 
                     args[16].dat->size[1] * 
                     (start_indx[2] * args[16].stencil->stride[2]);
    p_a[16] = (char *)args[16].data_d + base16;

    long long int base17 = args[17].dat->base_offset + dat17 * 1 * (start_indx[0] * args[17].stencil->stride[0]);
    base17 = base17 + dat17 * 
                     args[17].dat->size[0] * 
                     (start_indx[1] * args[17].stencil->stride[1]);
    base17 = base17 + dat17 * 
                     args[17].dat->size[0] * 
                     args[17].dat->size[1] * 
                     (start_indx[2] * args[17].stencil->stride[2]);
    p_a[17] = (char *)args[17].data_d + base17;

    long long int base18 = args[18].dat->base_offset + dat18 * 1 * (start_indx[0] * args[18].stencil->stride[0]);
    base18 = base18 + dat18 * 
                     args[18].dat->size[0] * 
                     (start_indx[1] * args[18].stencil->stride[1]);
    base18 = base18 + dat18 * 
                     args[18].dat->size[0] * 
                     args[18].dat->size[1] * 
                     (start_indx[2] * args[18].stencil->stride[2]);
    p_a[18] = (char *)args[18].data_d + base18;

    long long int base19 = args[19].dat->base_offset + dat19 * 1 * (start_indx[0] * args[19].stencil->stride[0]);
    base19 = base19 + dat19 * 
                     args[19].dat->size[0] * 
                     (start_indx[1] * args[19].stencil->stride[1]);
    base19 = base19 + dat19 * 
                     args[19].dat->size[0] * 
                     args[19].dat->size[1] * 
                     (start_indx[2] * args[19].stencil->stride[2]);
    p_a[19] = (char *)args[19].data_d + base19;

    long long int base20 = args[20].dat->base_offset + dat20 * 1 * (start_indx[0] * args[20].stencil->stride[0]);
    base20 = base20 + dat20 * 
                     args[20].dat->size[0] * 
                     (start_indx[1] * args[20].stencil->stride[1]);
    base20 = base20 + dat20 * 
                     args[20].dat->size[0] * 
                     args[20].dat->size[1] * 
                     (start_indx[2] * args[20].stencil->stride[2]);
    p_a[20] = (char *)args[20].data_d + base20;

    long long int base21 = args[21].dat->base_offset + dat21 * 1 * (start_indx[0] * args[21].stencil->stride[0]);
    base21 = base21 + dat21 * 
                     args[21].dat->size[0] * 
                     (start_indx[1] * args[21].stencil->stride[1]);
    base21 = base21 + dat21 * 
                     args[21].dat->size[0] * 
                     args[21].dat->size[1] * 
                     (start_indx[2] * args[21].stencil->stride[2]);
    p_a[21] = (char *)args[21].data_d + base21;

    long long int base22 = args[22].dat->base_offset + dat22 * 1 * (start_indx[0] * args[22].stencil->stride[0]);
    base22 = base22 + dat22 * 
                     args[22].dat->size[0] * 
                     (start_indx[1] * args[22].stencil->stride[1]);
    base22 = base22 + dat22 * 
                     args[22].dat->size[0] * 
                     args[22].dat->size[1] * 
                     (start_indx[2] * args[22].stencil->stride[2]);
    p_a[22] = (char *)args[22].data_d + base22;

    long long int base23 = args[23].dat->base_offset + dat23 * 1 * (start_indx[0] * args[23].stencil->stride[0]);
    base23 = base23 + dat23 * 
                     args[23].dat->size[0] * 
                     (start_indx[1] * args[23].stencil->stride[1]);
    base23 = base23 + dat23 * 
                     args[23].dat->size[0] * 
                     args[23].dat->size[1] * 
                     (start_indx[2] * args[23].stencil->stride[2]);
    p_a[23] = (char *)args[23].data_d + base23;

    long long int base24 = args[24].dat->base_offset + dat24 * 1 * (start_indx[0] * args[24].stencil->stride[0]);
    base24 = base24 + dat24 * 
                     args[24].dat->size[0] * 
                     (start_indx[1] * args[24].stencil->stride[1]);
    base24 = base24 + dat24 * 
                     args[24].dat->size[0] * 
                     args[24].dat->size[1] * 
                     (start_indx[2] * args[24].stencil->stride[2]);
    p_a[24] = (char *)args[24].data_d + base24;

    long long int base25 = args[25].dat->base_offset + dat25 * 1 * (start_indx[0] * args[25].stencil->stride[0]);
    base25 = base25 + dat25 * 
                     args[25].dat->size[0] * 
                     (start_indx[1] * args[25].stencil->stride[1]);
    base25 = base25 + dat25 * 
                     args[25].dat->size[0] * 
                     args[25].dat->size[1] * 
                     (start_indx[2] * args[25].stencil->stride[2]);
    p_a[25] = (char *)args[25].data_d + base25;

    long long int base26 = args[26].dat->base_offset + dat26 * 1 * (start_indx[0] * args[26].stencil->stride[0]);
    base26 = base26 + dat26 * 
                     args[26].dat->size[0] * 
                     (start_indx[1] * args[26].stencil->stride[1]);
    base26 = base26 + dat26 * 
                     args[26].dat->size[0] * 
                     args[26].dat->size[1] * 
                     (start_indx[2] * args[26].stencil->stride[2]);
    p_a[26] = (char *)args[26].data_d + base26;

    long long int base27 = args[27].dat->base_offset + dat27 * 1 * (start_indx[0] * args[27].stencil->stride[0]);
    base27 = base27 + dat27 * 
                     args[27].dat->size[0] * 
                     (start_indx[1] * args[27].stencil->stride[1]);
    base27 = base27 + dat27 * 
                     args[27].dat->size[0] * 
                     args[27].dat->size[1] * 
                     (start_indx[2] * args[27].stencil->stride[2]);
    p_a[27] = (char *)args[27].data_d + base27;

    long long int base28 = args[28].dat->base_offset + dat28 * 1 * (start_indx[0] * args[28].stencil->stride[0]);
    base28 = base28 + dat28 * 
                     args[28].dat->size[0] * 
                     (start_indx[1] * args[28].stencil->stride[1]);
    base28 = base28 + dat28 * 
                     args[28].dat->size[0] * 
                     args[28].dat->size[1] * 
                     (start_indx[2] * args[28].stencil->stride[2]);
    p_a[28] = (char *)args[28].data_d + base28;

    long long int base29 = args[29].dat->base_offset + dat29 * 1 * (start_indx[0] * args[29].stencil->stride[0]);
    base29 = base29 + dat29 * 
                     args[29].dat->size[0] * 
                     (start_indx[1] * args[29].stencil->stride[1]);
    base29 = base29 + dat29 * 
                     args[29].dat->size[0] * 
                     args[29].dat->size[1] * 
                     (start_indx[2] * args[29].stencil->stride[2]);
    p_a[29] = (char *)args[29].data_d + base29;

    long long int base30 = args[30].dat->base_offset + dat30 * 1 * (start_indx[0] * args[30].stencil->stride[0]);
    base30 = base30 + dat30 * 
                     args[30].dat->size[0] * 
                     (start_indx[1] * args[30].stencil->stride[1]);
    base30 = base30 + dat30 * 
                     args[30].dat->size[0] * 
                     args[30].dat->size[1] * 
                     (start_indx[2] * args[30].stencil->stride[2]);
    p_a[30] = (char *)args[30].data_d + base30;

    long long int base31 = args[31].dat->base_offset + dat31 * 1 * (start_indx[0] * args[31].stencil->stride[0]);
    base31 = base31 + dat31 * 
                     args[31].dat->size[0] * 
                     (start_indx[1] * args[31].stencil->stride[1]);
    base31 = base31 + dat31 * 
                     args[31].dat->size[0] * 
                     args[31].dat->size[1] * 
                     (start_indx[2] * args[31].stencil->stride[2]);
    p_a[31] = (char *)args[31].data_d + base31;

    long long int base32 = args[32].dat->base_offset + dat32 * 1 * (start_indx[0] * args[32].stencil->stride[0]);
    base32 = base32 + dat32 * 
                     args[32].dat->size[0] * 
                     (start_indx[1] * args[32].stencil->stride[1]);
    base32 = base32 + dat32 * 
                     args[32].dat->size[0] * 
                     args[32].dat->size[1] * 
                     (start_indx[2] * args[32].stencil->stride[2]);
    p_a[32] = (char *)args[32].data_d + base32;

//  =============
//  Halo exchange
//  =============
#ifndef OPS_LAZY
    ops_H_D_exchanges_device(args, 33);
    ops_halo_exchanges(args, 33, range);
#endif

    if (block->instance->OPS_diags > 1) { 
        ops_timers_core(&__c2, &__t2);
        block->instance->OPS_kernels[4].mpi_time += __t2 - __t1;
    }

//  ==========================================================
//  ops_dat strides for offset calculation in wrapper function
//  ==========================================================
    int xstride_7, ystride_7, zstride_7;
    xstride_7 = args[7].stencil->stride[0];    ystride_7 = args[7].stencil->stride[1];
    zstride_7 = args[7].stencil->stride[2];
    int xstride_8, ystride_8, zstride_8;
    xstride_8 = args[8].stencil->stride[0];    ystride_8 = args[8].stencil->stride[1];
    zstride_8 = args[8].stencil->stride[2];
    int xstride_9, ystride_9, zstride_9;
    xstride_9 = args[9].stencil->stride[0];    ystride_9 = args[9].stencil->stride[1];
    zstride_9 = args[9].stencil->stride[2];
    int xstride_10, ystride_10, zstride_10;
    xstride_10 = args[10].stencil->stride[0];    ystride_10 = args[10].stencil->stride[1];
    zstride_10 = args[10].stencil->stride[2];
    int xstride_11, ystride_11, zstride_11;
    xstride_11 = args[11].stencil->stride[0];    ystride_11 = args[11].stencil->stride[1];
    zstride_11 = args[11].stencil->stride[2];
    int xstride_12, ystride_12, zstride_12;
    xstride_12 = args[12].stencil->stride[0];    ystride_12 = args[12].stencil->stride[1];
    zstride_12 = args[12].stencil->stride[2];
    int xstride_13, ystride_13, zstride_13;
    xstride_13 = args[13].stencil->stride[0];    ystride_13 = args[13].stencil->stride[1];
    zstride_13 = args[13].stencil->stride[2];
    int xstride_14, ystride_14, zstride_14;
    xstride_14 = args[14].stencil->stride[0];    ystride_14 = args[14].stencil->stride[1];
    zstride_14 = args[14].stencil->stride[2];
    int xstride_15, ystride_15, zstride_15;
    xstride_15 = args[15].stencil->stride[0];    ystride_15 = args[15].stencil->stride[1];
    zstride_15 = args[15].stencil->stride[2];
    int xstride_16, ystride_16, zstride_16;
    xstride_16 = args[16].stencil->stride[0];    ystride_16 = args[16].stencil->stride[1];
    zstride_16 = args[16].stencil->stride[2];
    int xstride_17, ystride_17, zstride_17;
    xstride_17 = args[17].stencil->stride[0];    ystride_17 = args[17].stencil->stride[1];
    zstride_17 = args[17].stencil->stride[2];
    int xstride_18, ystride_18, zstride_18;
    xstride_18 = args[18].stencil->stride[0];    ystride_18 = args[18].stencil->stride[1];
    zstride_18 = args[18].stencil->stride[2];
    int xstride_19, ystride_19, zstride_19;
    xstride_19 = args[19].stencil->stride[0];    ystride_19 = args[19].stencil->stride[1];
    zstride_19 = args[19].stencil->stride[2];
    int xstride_20, ystride_20, zstride_20;
    xstride_20 = args[20].stencil->stride[0];    ystride_20 = args[20].stencil->stride[1];
    zstride_20 = args[20].stencil->stride[2];
    int xstride_21, ystride_21, zstride_21;
    xstride_21 = args[21].stencil->stride[0];    ystride_21 = args[21].stencil->stride[1];
    zstride_21 = args[21].stencil->stride[2];
    int xstride_22, ystride_22, zstride_22;
    xstride_22 = args[22].stencil->stride[0];    ystride_22 = args[22].stencil->stride[1];
    zstride_22 = args[22].stencil->stride[2];
    int xstride_23, ystride_23, zstride_23;
    xstride_23 = args[23].stencil->stride[0];    ystride_23 = args[23].stencil->stride[1];
    zstride_23 = args[23].stencil->stride[2];
    int xstride_24, ystride_24, zstride_24;
    xstride_24 = args[24].stencil->stride[0];    ystride_24 = args[24].stencil->stride[1];
    zstride_24 = args[24].stencil->stride[2];
    int xstride_25, ystride_25, zstride_25;
    xstride_25 = args[25].stencil->stride[0];    ystride_25 = args[25].stencil->stride[1];
    zstride_25 = args[25].stencil->stride[2];
    int xstride_26, ystride_26, zstride_26;
    xstride_26 = args[26].stencil->stride[0];    ystride_26 = args[26].stencil->stride[1];
    zstride_26 = args[26].stencil->stride[2];
    int xstride_27, ystride_27, zstride_27;
    xstride_27 = args[27].stencil->stride[0];    ystride_27 = args[27].stencil->stride[1];
    zstride_27 = args[27].stencil->stride[2];
    int xstride_28, ystride_28, zstride_28;
    xstride_28 = args[28].stencil->stride[0];    ystride_28 = args[28].stencil->stride[1];
    zstride_28 = args[28].stencil->stride[2];
    int xstride_29, ystride_29, zstride_29;
    xstride_29 = args[29].stencil->stride[0];    ystride_29 = args[29].stencil->stride[1];
    zstride_29 = args[29].stencil->stride[2];
    int xstride_30, ystride_30, zstride_30;
    xstride_30 = args[30].stencil->stride[0];    ystride_30 = args[30].stencil->stride[1];
    zstride_30 = args[30].stencil->stride[2];
    int xstride_31, ystride_31, zstride_31;
    xstride_31 = args[31].stencil->stride[0];    ystride_31 = args[31].stencil->stride[1];
    zstride_31 = args[31].stencil->stride[2];
    int xstride_32, ystride_32, zstride_32;
    xstride_32 = args[32].stencil->stride[0];    ystride_32 = args[32].stencil->stride[1];
    zstride_32 = args[32].stencil->stride[2];

//  call kernel wrapper function, passing in pointers to data
    if (x_size > 0 && y_size > 0 && z_size > 0) {

        ops_fd3d_pml_kernel2<<<grid, tblock >>> (
                   *(int *)arg0.data, 
                   *(int *)arg1.data, 
                   *(int *)arg2.data, 
                   arg_idx[0], arg_idx[1], arg_idx[2], 
                   *(float *)arg4.data, 
                   *(float *)arg5.data, 
                   *(float *)arg6.data, 
                   (float *)p_a[7], xstride_7, ystride_7, zstride_7, 
                   (float *)p_a[8], xstride_8, ystride_8, zstride_8, 
                   (float *)p_a[9], xstride_9, ystride_9, zstride_9, 
                   (float *)p_a[10], xstride_10, ystride_10, zstride_10, 
                   (float *)p_a[11], xstride_11, ystride_11, zstride_11, 
                   (float *)p_a[12], xstride_12, ystride_12, zstride_12, 
                   (float *)p_a[13], xstride_13, ystride_13, zstride_13, 
                   (float *)p_a[14], xstride_14, ystride_14, zstride_14, 
                   (float *)p_a[15], xstride_15, ystride_15, zstride_15, 
                   (float *)p_a[16], xstride_16, ystride_16, zstride_16, 
                   (float *)p_a[17], xstride_17, ystride_17, zstride_17, 
                   (float *)p_a[18], xstride_18, ystride_18, zstride_18, 
                   (float *)p_a[19], xstride_19, ystride_19, zstride_19, 
                   (float *)p_a[20], xstride_20, ystride_20, zstride_20, 
                   (float *)p_a[21], xstride_21, ystride_21, zstride_21, 
                   (float *)p_a[22], xstride_22, ystride_22, zstride_22, 
                   (float *)p_a[23], xstride_23, ystride_23, zstride_23, 
                   (float *)p_a[24], xstride_24, ystride_24, zstride_24, 
                   (float *)p_a[25], xstride_25, ystride_25, zstride_25, 
                   (float *)p_a[26], xstride_26, ystride_26, zstride_26, 
                   (float *)p_a[27], xstride_27, ystride_27, zstride_27, 
                   (float *)p_a[28], xstride_28, ystride_28, zstride_28, 
                   (float *)p_a[29], xstride_29, ystride_29, zstride_29, 
                   (float *)p_a[30], xstride_30, ystride_30, zstride_30, 
                   (float *)p_a[31], xstride_31, ystride_31, zstride_31, 
                   (float *)p_a[32], xstride_32, ystride_32, zstride_32, 
                   x_size, y_size, z_size);

    }

    cutilSafeCall(block->instance->ostream(), hipGetLastError());

    if(block->instance->OPS_diags > 1) {
        cutilSafeCall(block->instance->ostream(), hipDeviceSynchronize());
        ops_timers_core(&__c1, &__t1);
        block->instance->OPS_kernels[4].time += __t1 - __t2;
    }

#ifndef OPS_LAZY
    ops_set_dirtybit_device(args, 33);
    ops_set_halo_dirtybit3(&args[21], range);
    ops_set_halo_dirtybit3(&args[22], range);
    ops_set_halo_dirtybit3(&args[23], range);
    ops_set_halo_dirtybit3(&args[24], range);
    ops_set_halo_dirtybit3(&args[25], range);
    ops_set_halo_dirtybit3(&args[26], range);
    ops_set_halo_dirtybit3(&args[27], range);
    ops_set_halo_dirtybit3(&args[28], range);
    ops_set_halo_dirtybit3(&args[29], range);
    ops_set_halo_dirtybit3(&args[30], range);
    ops_set_halo_dirtybit3(&args[31], range);
    ops_set_halo_dirtybit3(&args[32], range);
#endif

    if (block->instance->OPS_diags > 1) {
//      ====================
//      Update kernel record
//      ====================
        ops_timers_core(&__c2, &__t2);
        block->instance->OPS_kernels[4].mpi_time += __t2 - __t1;
        block->instance->OPS_kernels[4].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg7);
        block->instance->OPS_kernels[4].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg8);
        block->instance->OPS_kernels[4].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg9);
        block->instance->OPS_kernels[4].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg10);
        block->instance->OPS_kernels[4].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg11);
        block->instance->OPS_kernels[4].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg12);
        block->instance->OPS_kernels[4].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg13);
        block->instance->OPS_kernels[4].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg14);
        block->instance->OPS_kernels[4].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg15);
        block->instance->OPS_kernels[4].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg16);
        block->instance->OPS_kernels[4].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg17);
        block->instance->OPS_kernels[4].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg18);
        block->instance->OPS_kernels[4].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg19);
        block->instance->OPS_kernels[4].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg20);
        block->instance->OPS_kernels[4].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg21);
        block->instance->OPS_kernels[4].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg22);
        block->instance->OPS_kernels[4].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg23);
        block->instance->OPS_kernels[4].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg24);
        block->instance->OPS_kernels[4].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg25);
        block->instance->OPS_kernels[4].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg26);
        block->instance->OPS_kernels[4].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg27);
        block->instance->OPS_kernels[4].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg28);
        block->instance->OPS_kernels[4].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg29);
        block->instance->OPS_kernels[4].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg30);
        block->instance->OPS_kernels[4].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg31);
        block->instance->OPS_kernels[4].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg32);
    }
}

#ifdef OPS_LAZY
void ops_par_loop_fd3d_pml_kernel2(
    const char * name,
    ops_block block,
    int dim,
    int * range,
    ops_arg arg0,
    ops_arg arg1,
    ops_arg arg2,
    ops_arg arg3,
    ops_arg arg4,
    ops_arg arg5,
    ops_arg arg6,
    ops_arg arg7,
    ops_arg arg8,
    ops_arg arg9,
    ops_arg arg10,
    ops_arg arg11,
    ops_arg arg12,
    ops_arg arg13,
    ops_arg arg14,
    ops_arg arg15,
    ops_arg arg16,
    ops_arg arg17,
    ops_arg arg18,
    ops_arg arg19,
    ops_arg arg20,
    ops_arg arg21,
    ops_arg arg22,
    ops_arg arg23,
    ops_arg arg24,
    ops_arg arg25,
    ops_arg arg26,
    ops_arg arg27,
    ops_arg arg28,
    ops_arg arg29,
    ops_arg arg30,
    ops_arg arg31,
    ops_arg arg32
    )
{
    ops_arg args[33];

    args[0] = arg0;
    args[1] = arg1;
    args[2] = arg2;
    args[3] = arg3;
    args[4] = arg4;
    args[5] = arg5;
    args[6] = arg6;
    args[7] = arg7;
    args[8] = arg8;
    args[9] = arg9;
    args[10] = arg10;
    args[11] = arg11;
    args[12] = arg12;
    args[13] = arg13;
    args[14] = arg14;
    args[15] = arg15;
    args[16] = arg16;
    args[17] = arg17;
    args[18] = arg18;
    args[19] = arg19;
    args[20] = arg20;
    args[21] = arg21;
    args[22] = arg22;
    args[23] = arg23;
    args[24] = arg24;
    args[25] = arg25;
    args[26] = arg26;
    args[27] = arg27;
    args[28] = arg28;
    args[29] = arg29;
    args[30] = arg30;
    args[31] = arg31;
    args[32] = arg32;

    create_kerneldesc_and_enque(name, "fd3d_pml_kernel2", args, 33, 4, dim, 1, range, block, ops_par_loop_fd3d_pml_kernel2_execute);
}
#endif
