#include "hip/hip_runtime.h"
// Auto-generated at 2025-08-06 23:23:48.283617 by ops-translator

__constant__ int dims_fd3d_pml_kernel1[27][2];
static int dims_fd3d_pml_kernel1_h[27][2] = {{0}};

//  =============
//  User function
//  =============
__device__ void fd3d_pml_kernel1_gpu(const int *dispx, const int *dispy, const int *dispz, const int *idx, const float *dt, const float *scale1, const float *scale2, const ACC<float> &rho, const ACC<float> &mu, const ACC<float> &yy_0, const ACC<float> &yy_1, const ACC<float> &yy_2, const ACC<float> &yy_3, const ACC<float> &yy_4, const ACC<float> &yy_5, ACC<float> &dyy_0, ACC<float> &dyy_1, ACC<float> &dyy_2, ACC<float> &dyy_3, ACC<float> &dyy_4, ACC<float> &dyy_5, ACC<float> &sum_0, ACC<float> &sum_1, ACC<float> &sum_2, ACC<float> &sum_3, ACC<float> &sum_4, ACC<float> &sum_5) {

    const float c[9] = {0.0035714285714285713,-0.0380952380952381,0.2,-0.8,0.0,0.8,-0.2,0.0380952380952381,-0.0035714285714285713};

    int xbeg=half_order;
    int xend=nx-half_order;
    int ybeg=half_order;
    int yend=ny-half_order;
    int zbeg=half_order;
    int zend=nz-half_order;
    int xpmlbeg=xbeg+pml_width;
    int ypmlbeg=ybeg+pml_width;
    int zpmlbeg=zbeg+pml_width;
    int xpmlend=xend-pml_width;
    int ypmlend=yend-pml_width;
    int zpmlend=zend-pml_width;

    float sigma = mu(0,0,0)/rho(0,0,0);
    float sigmax=0.0;
    float sigmay=0.0;
    float sigmaz=0.0;
    float sigma_factored = sigma * 0.1f;

    if(idx[0]<=xbeg+pml_width){
        float tmp0 = xbeg+pml_width;
        float tmp1 = tmp0 -idx[0];
        sigmax = tmp1 * sigma_factored;
    }
    if(idx[0]>=xend-pml_width){
        float tmp0 = xend-pml_width;
        float tmp1 = idx[0] - tmp0;
        sigmax = tmp1 * sigma_factored;
    }
    if(idx[1]<=ybeg+pml_width){
        float tmp0 = ybeg+pml_width;
        float tmp1 = tmp0 - idx[1];
        sigmay= tmp1 * sigma_factored;
    }
    if(idx[1]>=yend-pml_width){
        float tmp0 = yend-pml_width;
        float tmp1 = idx[1] - tmp0;
        sigmay= tmp1 * sigma_factored;
    }
    if(idx[2]<=zbeg+pml_width){
        float tmp0 = zbeg+pml_width;
        float tmp1 = tmp0 - idx[2];
        sigmaz=tmp1 * sigma_factored;
    }
    if(idx[2]>=zend-pml_width){
        float tmp0 = zend-pml_width;
        float tmp1 = idx[2] - tmp0;
        sigmaz=tmp1 * sigma_factored;
    }

    float px = yy_0(0,0,0);
    float py = yy_1(0,0,0);
    float pz = yy_2(0,0,0);

    float vx = yy_3(0,0,0);
    float vy = yy_4(0,0,0);
    float vz = yy_5(0,0,0);

    float vxx=0.0;
    float vxy=0.0;
    float vxz=0.0;

    float vyx=0.0;
    float vyy=0.0;
    float vyz=0.0;

    float vzx=0.0;
    float vzy=0.0;
    float vzz=0.0;

    float pxx=0.0;
    float pxy=0.0;
    float pxz=0.0;

    float pyx=0.0;
    float pyy=0.0;
    float pyz=0.0;

    float pzx=0.0;
    float pzy=0.0;
    float pzz=0.0;

    pxx += yy_0(-4,0,0)*c[-4+half_order];
    pyx += yy_1(-4,0,0)*c[-4+half_order];
    pzx += yy_2(-4,0,0)*c[-4+half_order];

    vxx += yy_3(-4,0,0)*c[-4+half_order];
    vyx += yy_4(-4,0,0)*c[-4+half_order];
    vzx += yy_5(-4,0,0)*c[-4+half_order];

    pxy += yy_0(0,-4,0)*c[-4+half_order];
    pyy += yy_1(0,-4,0)*c[-4+half_order];
    pzy += yy_2(0,-4,0)*c[-4+half_order];

    vxy += yy_3(0,-4,0)*c[-4+half_order];
    vyy += yy_4(0,-4,0)*c[-4+half_order];
    vzy += yy_5(0,-4,0)*c[-4+half_order];

    pxz += yy_0(0,0,-4)*c[-4+half_order];
    pyz += yy_1(0,0,-4)*c[-4+half_order];
    pzz += yy_2(0,0,-4)*c[-4+half_order];

    vxz += yy_3(0,0,-4)*c[-4+half_order];
    vyz += yy_4(0,0,-4)*c[-4+half_order];
    vzz += yy_5(0,0,-4)*c[-4+half_order];

    pxx += yy_0(-3,0,0)*c[-3+half_order];
    pyx += yy_1(-3,0,0)*c[-3+half_order];
    pzx += yy_2(-3,0,0)*c[-3+half_order];

    vxx += yy_3(-3,0,0)*c[-3+half_order];
    vyx += yy_4(-3,0,0)*c[-3+half_order];
    vzx += yy_5(-3,0,0)*c[-3+half_order];

    pxy += yy_0(0,-3,0)*c[-3+half_order];
    pyy += yy_1(0,-3,0)*c[-3+half_order];
    pzy += yy_2(0,-3,0)*c[-3+half_order];

    vxy += yy_3(0,-3,0)*c[-3+half_order];
    vyy += yy_4(0,-3,0)*c[-3+half_order];
    vzy += yy_5(0,-3,0)*c[-3+half_order];

    pxz += yy_0(0,0,-3)*c[-3+half_order];
    pyz += yy_1(0,0,-3)*c[-3+half_order];
    pzz += yy_2(0,0,-3)*c[-3+half_order];

    vxz += yy_3(0,0,-3)*c[-3+half_order];
    vyz += yy_4(0,0,-3)*c[-3+half_order];
    vzz += yy_5(0,0,-3)*c[-3+half_order];

    pxx += yy_0(-2,0,0)*c[-2+half_order];
    pyx += yy_1(-2,0,0)*c[-2+half_order];
    pzx += yy_2(-2,0,0)*c[-2+half_order];

    vxx += yy_3(-2,0,0)*c[-2+half_order];
    vyx += yy_4(-2,0,0)*c[-2+half_order];
    vzx += yy_5(-2,0,0)*c[-2+half_order];

    pxy += yy_0(0,-2,0)*c[-2+half_order];
    pyy += yy_1(0,-2,0)*c[-2+half_order];
    pzy += yy_2(0,-2,0)*c[-2+half_order];

    vxy += yy_3(0,-2,0)*c[-2+half_order];
    vyy += yy_4(0,-2,0)*c[-2+half_order];
    vzy += yy_5(0,-2,0)*c[-2+half_order];

    pxz += yy_0(0,0,-2)*c[-2+half_order];
    pyz += yy_1(0,0,-2)*c[-2+half_order];
    pzz += yy_2(0,0,-2)*c[-2+half_order];

    vxz += yy_3(0,0,-2)*c[-2+half_order];
    vyz += yy_4(0,0,-2)*c[-2+half_order];
    vzz += yy_5(0,0,-2)*c[-2+half_order];

    pxx += yy_0(-1,0,0)*c[-1+half_order];
    pyx += yy_1(-1,0,0)*c[-1+half_order];
    pzx += yy_2(-1,0,0)*c[-1+half_order];

    vxx += yy_3(-1,0,0)*c[-1+half_order];
    vyx += yy_4(-1,0,0)*c[-1+half_order];
    vzx += yy_5(-1,0,0)*c[-1+half_order];

    pxy += yy_0(0,-1,0)*c[-1+half_order];
    pyy += yy_1(0,-1,0)*c[-1+half_order];
    pzy += yy_2(0,-1,0)*c[-1+half_order];

    vxy += yy_3(0,-1,0)*c[-1+half_order];
    vyy += yy_4(0,-1,0)*c[-1+half_order];
    vzy += yy_5(0,-1,0)*c[-1+half_order];

    pxz += yy_0(0,0,-1)*c[-1+half_order];
    pyz += yy_1(0,0,-1)*c[-1+half_order];
    pzz += yy_2(0,0,-1)*c[-1+half_order];

    vxz += yy_3(0,0,-1)*c[-1+half_order];
    vyz += yy_4(0,0,-1)*c[-1+half_order];
    vzz += yy_5(0,0,-1)*c[-1+half_order];

    pxx += yy_0(0,0,0)*c[half_order];
    pyx += yy_1(0,0,0)*c[half_order];
    pzx += yy_2(0,0,0)*c[half_order];

    vxx += yy_3(0,0,0)*c[half_order];
    vyx += yy_4(0,0,0)*c[half_order];
    vzx += yy_5(0,0,0)*c[half_order];

    pxy += yy_0(0,0,0)*c[half_order];
    pyy += yy_1(0,0,0)*c[half_order];
    pzy += yy_2(0,0,0)*c[half_order];

    vxy += yy_3(0,0,0)*c[half_order];
    vyy += yy_4(0,0,0)*c[half_order];
    vzy += yy_5(0,0,0)*c[half_order];

    pxz += yy_0(0,0,0)*c[half_order];
    pyz += yy_1(0,0,0)*c[half_order];
    pzz += yy_2(0,0,0)*c[half_order];

    vxz += yy_3(0,0,0)*c[half_order];
    vyz += yy_4(0,0,0)*c[half_order];
    vzz += yy_5(0,0,0)*c[half_order];

    pxx += yy_0(1,0,0)*c[1+half_order];
    pyx += yy_1(1,0,0)*c[1+half_order];
    pzx += yy_2(1,0,0)*c[1+half_order];

    vxx += yy_3(1,0,0)*c[1+half_order];
    vyx += yy_4(1,0,0)*c[1+half_order];
    vzx += yy_5(1,0,0)*c[1+half_order];

    pxy += yy_0(0,1,0)*c[1+half_order];
    pyy += yy_1(0,1,0)*c[1+half_order];
    pzy += yy_2(0,1,0)*c[1+half_order];

    vxy += yy_3(0,1,0)*c[1+half_order];
    vyy += yy_4(0,1,0)*c[1+half_order];
    vzy += yy_5(0,1,0)*c[1+half_order];

    pxz += yy_0(0,0,1)*c[1+half_order];
    pyz += yy_1(0,0,1)*c[1+half_order];
    pzz += yy_2(0,0,1)*c[1+half_order];

    vxz += yy_3(0,0,1)*c[1+half_order];
    vyz += yy_4(0,0,1)*c[1+half_order];
    vzz += yy_5(0,0,1)*c[1+half_order];

    pxx += yy_0(2,0,0)*c[2+half_order];
    pyx += yy_1(2,0,0)*c[2+half_order];
    pzx += yy_2(2,0,0)*c[2+half_order];

    vxx += yy_3(2,0,0)*c[2+half_order];
    vyx += yy_4(2,0,0)*c[2+half_order];
    vzx += yy_5(2,0,0)*c[2+half_order];

    pxy += yy_0(0,2,0)*c[2+half_order];
    pyy += yy_1(0,2,0)*c[2+half_order];
    pzy += yy_2(0,2,0)*c[2+half_order];

    vxy += yy_3(0,2,0)*c[2+half_order];
    vyy += yy_4(0,2,0)*c[2+half_order];
    vzy += yy_5(0,2,0)*c[2+half_order];

    pxz += yy_0(0,0,2)*c[2+half_order];
    pyz += yy_1(0,0,2)*c[2+half_order];
    pzz += yy_2(0,0,2)*c[2+half_order];

    vxz += yy_3(0,0,2)*c[2+half_order];
    vyz += yy_4(0,0,2)*c[2+half_order];
    vzz += yy_5(0,0,2)*c[2+half_order];

    pxx += yy_0(3,0,0)*c[3+half_order];
    pyx += yy_1(3,0,0)*c[3+half_order];
    pzx += yy_2(3,0,0)*c[3+half_order];

    vxx += yy_3(3,0,0)*c[3+half_order];
    vyx += yy_4(3,0,0)*c[3+half_order];
    vzx += yy_5(3,0,0)*c[3+half_order];

    pxy += yy_0(0,3,0)*c[3+half_order];
    pyy += yy_1(0,3,0)*c[3+half_order];
    pzy += yy_2(0,3,0)*c[3+half_order];

    vxy += yy_3(0,3,0)*c[3+half_order];
    vyy += yy_4(0,3,0)*c[3+half_order];
    vzy += yy_5(0,3,0)*c[3+half_order];

    pxz += yy_0(0,0,3)*c[3+half_order];
    pyz += yy_1(0,0,3)*c[3+half_order];
    pzz += yy_2(0,0,3)*c[3+half_order];

    vxz += yy_3(0,0,3)*c[3+half_order];
    vyz += yy_4(0,0,3)*c[3+half_order];
    vzz += yy_5(0,0,3)*c[3+half_order];

    pxx += yy_0(4,0,0)*c[4+half_order];
    pyx += yy_1(4,0,0)*c[4+half_order];
    pzx += yy_2(4,0,0)*c[4+half_order];

    vxx += yy_3(4,0,0)*c[4+half_order];
    vyx += yy_4(4,0,0)*c[4+half_order];
    vzx += yy_5(4,0,0)*c[4+half_order];

    pxy += yy_0(0,4,0)*c[4+half_order];
    pyy += yy_1(0,4,0)*c[4+half_order];
    pzy += yy_2(0,4,0)*c[4+half_order];

    vxy += yy_3(0,4,0)*c[4+half_order];
    vyy += yy_4(0,4,0)*c[4+half_order];
    vzy += yy_5(0,4,0)*c[4+half_order];

    pxz += yy_0(0,0,4)*c[4+half_order];
    pyz += yy_1(0,0,4)*c[4+half_order];
    pzz += yy_2(0,0,4)*c[4+half_order];

    vxz += yy_3(0,0,4)*c[4+half_order];
    vyz += yy_4(0,0,4)*c[4+half_order];
    vzz += yy_5(0,0,4)*c[4+half_order];

    pxx *= invdx;
    pyx *= invdx;
    pzx *= invdx;

    vxx *= invdx;
    vyx *= invdx;
    vzx *= invdx;

    pxy *= invdy;
    pyy *= invdy;
    pzy *= invdy;

    vxy *= invdy;
    vyy *= invdy;
    vzy *= invdy;

    pxz *= invdz;
    pyz *= invdz;
    pzz *= invdz;

    vxz *= invdz;
    vyz *= invdz;
    vzz *= invdz;

    float vxx_div_rho = vxx/rho(0,0,0);
    float sigmax_px = sigmax*px;
    float sum_pxx_pyx_pxz = pxx+pyx+pxz;
    float sum_pxx_pyx_pxz_mu = sum_pxx_pyx_pxz * mu(0,0,0);
    float sigmax_vx = sigmax * vx;

    float ytemp0 =(vxx_div_rho - sigmax_px) * *dt;
    float ytemp3 =(sum_pxx_pyx_pxz_mu - sigmax_vx)* *dt;

    float vyy_div_rho = vyy/rho(0,0,0);
    float sigmay_py = sigmay * py;
    float sum_pxy_pyy_pyz = pxy+pyy+pyz;
    float sum_pxy_pyy_pyz_mu = sum_pxy_pyy_pyz * mu(0,0,0);
    float sigmay_vy = sigmay * vy;

    float ytemp1 =(vyy_div_rho - sigmay_py) * *dt;
    float ytemp4 =(sum_pxy_pyy_pyz_mu - sigmay_vy) * *dt;

    float vzz_div_rho = vzz/rho(0,0,0);
    float sigmaz_pz = sigmaz*pz;
    float sum_pxz_pyz_pzz = pxz+pyz+pzz;
    float sum_pxz_pyz_pzz_mu = sum_pxz_pyz_pzz * mu(0,0,0);
    float sigmaz_vz = sigmaz*vz;

    float ytemp2 =(vzz_div_rho - sigmaz_pz)* *dt;
    float ytemp5 =(sum_pxz_pyz_pzz_mu - sigmaz_vz)* *dt;

    float ytemp0_scale1 = ytemp0* *scale1;
    float ytemp1_scale1 = ytemp1* *scale1;
    float ytemp2_scale1 = ytemp2* *scale1;
    float ytemp3_scale1 = ytemp3* *scale1;
    float ytemp4_scale1 = ytemp4* *scale1;
    float ytemp5_scale1 = ytemp5* *scale1;

    dyy_0(0,0,0) = yy_0(0,0,0) + ytemp0_scale1;
    dyy_3(0,0,0) = yy_3(0,0,0) + ytemp3_scale1;
    dyy_1(0,0,0) = yy_1(0,0,0) + ytemp1_scale1;
    dyy_4(0,0,0) = yy_4(0,0,0) + ytemp4_scale1;
    dyy_2(0,0,0) = yy_2(0,0,0) + ytemp2_scale1;
    dyy_5(0,0,0) = yy_5(0,0,0) + ytemp5_scale1;

    float ytemp0_scale2 = ytemp0 * *scale2;
    float ytemp1_scale2 = ytemp1 * *scale2;
    float ytemp2_scale2 = ytemp2 * *scale2;
    float ytemp3_scale2 = ytemp3 * *scale2;
    float ytemp4_scale2 = ytemp4 * *scale2;
    float ytemp5_scale2 = ytemp5 * *scale2;

    sum_0(0,0,0) += ytemp0_scale2;
    sum_3(0,0,0) += ytemp3_scale2;
    sum_1(0,0,0) += ytemp1_scale2;
    sum_4(0,0,0) += ytemp4_scale2;
    sum_2(0,0,0) += ytemp2_scale2;
    sum_5(0,0,0) += ytemp5_scale2;

}

//  ============================
//  Cuda kernel wrapper function
//  ============================
__global__ void ops_fd3d_pml_kernel1(const int arg0, 
const int arg1, 
const int arg2, 
int arg_idx0, int arg_idx1, int arg_idx2, 
const float arg4, 
const float arg5, 
const float arg6, 
float* __restrict arg7, int xstride_7, int ystride_7, int zstride_7, 
float* __restrict arg8, int xstride_8, int ystride_8, int zstride_8, 
float* __restrict arg9, int xstride_9, int ystride_9, int zstride_9, 
float* __restrict arg10, int xstride_10, int ystride_10, int zstride_10, 
float* __restrict arg11, int xstride_11, int ystride_11, int zstride_11, 
float* __restrict arg12, int xstride_12, int ystride_12, int zstride_12, 
float* __restrict arg13, int xstride_13, int ystride_13, int zstride_13, 
float* __restrict arg14, int xstride_14, int ystride_14, int zstride_14, 
float* __restrict arg15, int xstride_15, int ystride_15, int zstride_15, 
float* __restrict arg16, int xstride_16, int ystride_16, int zstride_16, 
float* __restrict arg17, int xstride_17, int ystride_17, int zstride_17, 
float* __restrict arg18, int xstride_18, int ystride_18, int zstride_18, 
float* __restrict arg19, int xstride_19, int ystride_19, int zstride_19, 
float* __restrict arg20, int xstride_20, int ystride_20, int zstride_20, 
float* __restrict arg21, int xstride_21, int ystride_21, int zstride_21, 
float* __restrict arg22, int xstride_22, int ystride_22, int zstride_22, 
float* __restrict arg23, int xstride_23, int ystride_23, int zstride_23, 
float* __restrict arg24, int xstride_24, int ystride_24, int zstride_24, 
float* __restrict arg25, int xstride_25, int ystride_25, int zstride_25, 
float* __restrict arg26, int xstride_26, int ystride_26, int zstride_26, 
int size0, int size1, int size2) {

    int idx_z = blockDim.z * blockIdx.z + threadIdx.z;
    int idx_y = blockDim.y * blockIdx.y + threadIdx.y;
    int idx_x = blockDim.x * blockIdx.x + threadIdx.x;

    int arg_idx[3];
    arg_idx[0] = arg_idx0+idx_x;
    arg_idx[1] = arg_idx1+idx_y;
    arg_idx[2] = arg_idx2+idx_z;

    arg7 += idx_x * xstride_7*1 + idx_y * ystride_7*1 * dims_fd3d_pml_kernel1[7][0] + idx_z * zstride_7*1 * dims_fd3d_pml_kernel1[7][0] * dims_fd3d_pml_kernel1[7][1];
    arg8 += idx_x * xstride_8*1 + idx_y * ystride_8*1 * dims_fd3d_pml_kernel1[8][0] + idx_z * zstride_8*1 * dims_fd3d_pml_kernel1[8][0] * dims_fd3d_pml_kernel1[8][1];
    arg9 += idx_x * xstride_9*1 + idx_y * ystride_9*1 * dims_fd3d_pml_kernel1[9][0] + idx_z * zstride_9*1 * dims_fd3d_pml_kernel1[9][0] * dims_fd3d_pml_kernel1[9][1];
    arg10 += idx_x * xstride_10*1 + idx_y * ystride_10*1 * dims_fd3d_pml_kernel1[10][0] + idx_z * zstride_10*1 * dims_fd3d_pml_kernel1[10][0] * dims_fd3d_pml_kernel1[10][1];
    arg11 += idx_x * xstride_11*1 + idx_y * ystride_11*1 * dims_fd3d_pml_kernel1[11][0] + idx_z * zstride_11*1 * dims_fd3d_pml_kernel1[11][0] * dims_fd3d_pml_kernel1[11][1];
    arg12 += idx_x * xstride_12*1 + idx_y * ystride_12*1 * dims_fd3d_pml_kernel1[12][0] + idx_z * zstride_12*1 * dims_fd3d_pml_kernel1[12][0] * dims_fd3d_pml_kernel1[12][1];
    arg13 += idx_x * xstride_13*1 + idx_y * ystride_13*1 * dims_fd3d_pml_kernel1[13][0] + idx_z * zstride_13*1 * dims_fd3d_pml_kernel1[13][0] * dims_fd3d_pml_kernel1[13][1];
    arg14 += idx_x * xstride_14*1 + idx_y * ystride_14*1 * dims_fd3d_pml_kernel1[14][0] + idx_z * zstride_14*1 * dims_fd3d_pml_kernel1[14][0] * dims_fd3d_pml_kernel1[14][1];
    arg15 += idx_x * xstride_15*1 + idx_y * ystride_15*1 * dims_fd3d_pml_kernel1[15][0] + idx_z * zstride_15*1 * dims_fd3d_pml_kernel1[15][0] * dims_fd3d_pml_kernel1[15][1];
    arg16 += idx_x * xstride_16*1 + idx_y * ystride_16*1 * dims_fd3d_pml_kernel1[16][0] + idx_z * zstride_16*1 * dims_fd3d_pml_kernel1[16][0] * dims_fd3d_pml_kernel1[16][1];
    arg17 += idx_x * xstride_17*1 + idx_y * ystride_17*1 * dims_fd3d_pml_kernel1[17][0] + idx_z * zstride_17*1 * dims_fd3d_pml_kernel1[17][0] * dims_fd3d_pml_kernel1[17][1];
    arg18 += idx_x * xstride_18*1 + idx_y * ystride_18*1 * dims_fd3d_pml_kernel1[18][0] + idx_z * zstride_18*1 * dims_fd3d_pml_kernel1[18][0] * dims_fd3d_pml_kernel1[18][1];
    arg19 += idx_x * xstride_19*1 + idx_y * ystride_19*1 * dims_fd3d_pml_kernel1[19][0] + idx_z * zstride_19*1 * dims_fd3d_pml_kernel1[19][0] * dims_fd3d_pml_kernel1[19][1];
    arg20 += idx_x * xstride_20*1 + idx_y * ystride_20*1 * dims_fd3d_pml_kernel1[20][0] + idx_z * zstride_20*1 * dims_fd3d_pml_kernel1[20][0] * dims_fd3d_pml_kernel1[20][1];
    arg21 += idx_x * xstride_21*1 + idx_y * ystride_21*1 * dims_fd3d_pml_kernel1[21][0] + idx_z * zstride_21*1 * dims_fd3d_pml_kernel1[21][0] * dims_fd3d_pml_kernel1[21][1];
    arg22 += idx_x * xstride_22*1 + idx_y * ystride_22*1 * dims_fd3d_pml_kernel1[22][0] + idx_z * zstride_22*1 * dims_fd3d_pml_kernel1[22][0] * dims_fd3d_pml_kernel1[22][1];
    arg23 += idx_x * xstride_23*1 + idx_y * ystride_23*1 * dims_fd3d_pml_kernel1[23][0] + idx_z * zstride_23*1 * dims_fd3d_pml_kernel1[23][0] * dims_fd3d_pml_kernel1[23][1];
    arg24 += idx_x * xstride_24*1 + idx_y * ystride_24*1 * dims_fd3d_pml_kernel1[24][0] + idx_z * zstride_24*1 * dims_fd3d_pml_kernel1[24][0] * dims_fd3d_pml_kernel1[24][1];
    arg25 += idx_x * xstride_25*1 + idx_y * ystride_25*1 * dims_fd3d_pml_kernel1[25][0] + idx_z * zstride_25*1 * dims_fd3d_pml_kernel1[25][0] * dims_fd3d_pml_kernel1[25][1];
    arg26 += idx_x * xstride_26*1 + idx_y * ystride_26*1 * dims_fd3d_pml_kernel1[26][0] + idx_z * zstride_26*1 * dims_fd3d_pml_kernel1[26][0] * dims_fd3d_pml_kernel1[26][1];

    if(idx_x < size0 && idx_y < size1 && idx_z < size2) {

        const ACC<float> argp7(dims_fd3d_pml_kernel1[7][0], dims_fd3d_pml_kernel1[7][1], arg7);
        const ACC<float> argp8(dims_fd3d_pml_kernel1[8][0], dims_fd3d_pml_kernel1[8][1], arg8);
        const ACC<float> argp9(dims_fd3d_pml_kernel1[9][0], dims_fd3d_pml_kernel1[9][1], arg9);
        const ACC<float> argp10(dims_fd3d_pml_kernel1[10][0], dims_fd3d_pml_kernel1[10][1], arg10);
        const ACC<float> argp11(dims_fd3d_pml_kernel1[11][0], dims_fd3d_pml_kernel1[11][1], arg11);
        const ACC<float> argp12(dims_fd3d_pml_kernel1[12][0], dims_fd3d_pml_kernel1[12][1], arg12);
        const ACC<float> argp13(dims_fd3d_pml_kernel1[13][0], dims_fd3d_pml_kernel1[13][1], arg13);
        const ACC<float> argp14(dims_fd3d_pml_kernel1[14][0], dims_fd3d_pml_kernel1[14][1], arg14);
        ACC<float> argp15(dims_fd3d_pml_kernel1[15][0], dims_fd3d_pml_kernel1[15][1], arg15);
        ACC<float> argp16(dims_fd3d_pml_kernel1[16][0], dims_fd3d_pml_kernel1[16][1], arg16);
        ACC<float> argp17(dims_fd3d_pml_kernel1[17][0], dims_fd3d_pml_kernel1[17][1], arg17);
        ACC<float> argp18(dims_fd3d_pml_kernel1[18][0], dims_fd3d_pml_kernel1[18][1], arg18);
        ACC<float> argp19(dims_fd3d_pml_kernel1[19][0], dims_fd3d_pml_kernel1[19][1], arg19);
        ACC<float> argp20(dims_fd3d_pml_kernel1[20][0], dims_fd3d_pml_kernel1[20][1], arg20);
        ACC<float> argp21(dims_fd3d_pml_kernel1[21][0], dims_fd3d_pml_kernel1[21][1], arg21);
        ACC<float> argp22(dims_fd3d_pml_kernel1[22][0], dims_fd3d_pml_kernel1[22][1], arg22);
        ACC<float> argp23(dims_fd3d_pml_kernel1[23][0], dims_fd3d_pml_kernel1[23][1], arg23);
        ACC<float> argp24(dims_fd3d_pml_kernel1[24][0], dims_fd3d_pml_kernel1[24][1], arg24);
        ACC<float> argp25(dims_fd3d_pml_kernel1[25][0], dims_fd3d_pml_kernel1[25][1], arg25);
        ACC<float> argp26(dims_fd3d_pml_kernel1[26][0], dims_fd3d_pml_kernel1[26][1], arg26);

        fd3d_pml_kernel1_gpu(&arg0, &arg1, &arg2, arg_idx, &arg4, &arg5, &arg6, argp7, argp8, argp9, argp10, argp11, argp12, argp13, argp14, argp15, argp16, argp17, argp18, argp19, argp20, argp21, argp22, argp23, argp24, argp25, argp26);

    }// End of cuda index in_range check

}// End of cuda kernel wrapper function

//  ==================
//  Host stub function
//  ==================
#ifndef OPS_LAZY
void ops_par_loop_fd3d_pml_kernel1(
    const char * name,
    ops_block block,
    int dim,
    int *range,
    ops_arg arg0,
    ops_arg arg1,
    ops_arg arg2,
    ops_arg arg3,
    ops_arg arg4,
    ops_arg arg5,
    ops_arg arg6,
    ops_arg arg7,
    ops_arg arg8,
    ops_arg arg9,
    ops_arg arg10,
    ops_arg arg11,
    ops_arg arg12,
    ops_arg arg13,
    ops_arg arg14,
    ops_arg arg15,
    ops_arg arg16,
    ops_arg arg17,
    ops_arg arg18,
    ops_arg arg19,
    ops_arg arg20,
    ops_arg arg21,
    ops_arg arg22,
    ops_arg arg23,
    ops_arg arg24,
    ops_arg arg25,
    ops_arg arg26
)
{ 
#else
void ops_par_loop_fd3d_pml_kernel1_execute(ops_kernel_descriptor *desc)
{
    ops_block block = desc->block;
    int dim = desc->dim;
    int *range = desc->range;
    ops_arg arg0 = desc->args[0];
    ops_arg arg1 = desc->args[1];
    ops_arg arg2 = desc->args[2];
    ops_arg arg3 = desc->args[3];
    ops_arg arg4 = desc->args[4];
    ops_arg arg5 = desc->args[5];
    ops_arg arg6 = desc->args[6];
    ops_arg arg7 = desc->args[7];
    ops_arg arg8 = desc->args[8];
    ops_arg arg9 = desc->args[9];
    ops_arg arg10 = desc->args[10];
    ops_arg arg11 = desc->args[11];
    ops_arg arg12 = desc->args[12];
    ops_arg arg13 = desc->args[13];
    ops_arg arg14 = desc->args[14];
    ops_arg arg15 = desc->args[15];
    ops_arg arg16 = desc->args[16];
    ops_arg arg17 = desc->args[17];
    ops_arg arg18 = desc->args[18];
    ops_arg arg19 = desc->args[19];
    ops_arg arg20 = desc->args[20];
    ops_arg arg21 = desc->args[21];
    ops_arg arg22 = desc->args[22];
    ops_arg arg23 = desc->args[23];
    ops_arg arg24 = desc->args[24];
    ops_arg arg25 = desc->args[25];
    ops_arg arg26 = desc->args[26];
#endif

//  ======
//  Timing
//  ======
    double __t1, __t2, __c1, __c2;

    ops_arg args[27];

    args[0] = arg0;
    args[1] = arg1;
    args[2] = arg2;
    args[3] = arg3;
    args[4] = arg4;
    args[5] = arg5;
    args[6] = arg6;
    args[7] = arg7;
    args[8] = arg8;
    args[9] = arg9;
    args[10] = arg10;
    args[11] = arg11;
    args[12] = arg12;
    args[13] = arg13;
    args[14] = arg14;
    args[15] = arg15;
    args[16] = arg16;
    args[17] = arg17;
    args[18] = arg18;
    args[19] = arg19;
    args[20] = arg20;
    args[21] = arg21;
    args[22] = arg22;
    args[23] = arg23;
    args[24] = arg24;
    args[25] = arg25;
    args[26] = arg26;

#if defined(CHECKPOINTING) && !defined(OPS_LAZY)
    if (!ops_checkpointing_before(args, 27, range, 3)) return;
#endif

    if (block->instance->OPS_diags > 1)
    {
        ops_timing_realloc(block->instance, 3, "fd3d_pml_kernel1");
        block->instance->OPS_kernels[3].count++;
        ops_timers_core(&__c1, &__t1);
    }

//  =================================================
//  compute locally allocated range for the sub-block
//  =================================================
    int start_indx[3];
    int end_indx[3];
    int arg_idx[3];

#if defined(OPS_LAZY) || !defined(OPS_MPI)
    for (int n = 0; n < 3; n++) {
        start_indx[n] = range[2*n];
        end_indx[n]   = range[2*n+1];
    }
#else
    if (compute_ranges(args, 27, block, range, start_indx, end_indx, arg_idx) < 0) return;
#endif

#if defined(OPS_MPI)
#if defined(OPS_LAZY)
    sub_block_list sb = OPS_sub_block_list[block->index];
    arg_idx[0] = sb->decomp_disp[0]+start_indx[0];
    arg_idx[1] = sb->decomp_disp[1]+start_indx[1];
    arg_idx[2] = sb->decomp_disp[2]+start_indx[2];
#endif  // OPS_LAZY
#else   // OPS_MPI
    arg_idx[0] = start_indx[0];
    arg_idx[1] = start_indx[1];
    arg_idx[2] = start_indx[2];
#endif  // OPS_MPI

    int xdim7 = args[7].dat->size[0];
    int ydim7 = args[7].dat->size[1];
    int xdim8 = args[8].dat->size[0];
    int ydim8 = args[8].dat->size[1];
    int xdim9 = args[9].dat->size[0];
    int ydim9 = args[9].dat->size[1];
    int xdim10 = args[10].dat->size[0];
    int ydim10 = args[10].dat->size[1];
    int xdim11 = args[11].dat->size[0];
    int ydim11 = args[11].dat->size[1];
    int xdim12 = args[12].dat->size[0];
    int ydim12 = args[12].dat->size[1];
    int xdim13 = args[13].dat->size[0];
    int ydim13 = args[13].dat->size[1];
    int xdim14 = args[14].dat->size[0];
    int ydim14 = args[14].dat->size[1];
    int xdim15 = args[15].dat->size[0];
    int ydim15 = args[15].dat->size[1];
    int xdim16 = args[16].dat->size[0];
    int ydim16 = args[16].dat->size[1];
    int xdim17 = args[17].dat->size[0];
    int ydim17 = args[17].dat->size[1];
    int xdim18 = args[18].dat->size[0];
    int ydim18 = args[18].dat->size[1];
    int xdim19 = args[19].dat->size[0];
    int ydim19 = args[19].dat->size[1];
    int xdim20 = args[20].dat->size[0];
    int ydim20 = args[20].dat->size[1];
    int xdim21 = args[21].dat->size[0];
    int ydim21 = args[21].dat->size[1];
    int xdim22 = args[22].dat->size[0];
    int ydim22 = args[22].dat->size[1];
    int xdim23 = args[23].dat->size[0];
    int ydim23 = args[23].dat->size[1];
    int xdim24 = args[24].dat->size[0];
    int ydim24 = args[24].dat->size[1];
    int xdim25 = args[25].dat->size[0];
    int ydim25 = args[25].dat->size[1];
    int xdim26 = args[26].dat->size[0];
    int ydim26 = args[26].dat->size[1];

    if (xdim7 != dims_fd3d_pml_kernel1_h[7][0] || ydim7 != dims_fd3d_pml_kernel1_h[7][1] || xdim8 != dims_fd3d_pml_kernel1_h[8][0] || ydim8 != dims_fd3d_pml_kernel1_h[8][1] || xdim9 != dims_fd3d_pml_kernel1_h[9][0] || ydim9 != dims_fd3d_pml_kernel1_h[9][1] || xdim10 != dims_fd3d_pml_kernel1_h[10][0] || ydim10 != dims_fd3d_pml_kernel1_h[10][1] || xdim11 != dims_fd3d_pml_kernel1_h[11][0] || ydim11 != dims_fd3d_pml_kernel1_h[11][1] || xdim12 != dims_fd3d_pml_kernel1_h[12][0] || ydim12 != dims_fd3d_pml_kernel1_h[12][1] || xdim13 != dims_fd3d_pml_kernel1_h[13][0] || ydim13 != dims_fd3d_pml_kernel1_h[13][1] || xdim14 != dims_fd3d_pml_kernel1_h[14][0] || ydim14 != dims_fd3d_pml_kernel1_h[14][1] || xdim15 != dims_fd3d_pml_kernel1_h[15][0] || ydim15 != dims_fd3d_pml_kernel1_h[15][1] || xdim16 != dims_fd3d_pml_kernel1_h[16][0] || ydim16 != dims_fd3d_pml_kernel1_h[16][1] || xdim17 != dims_fd3d_pml_kernel1_h[17][0] || ydim17 != dims_fd3d_pml_kernel1_h[17][1] || xdim18 != dims_fd3d_pml_kernel1_h[18][0] || ydim18 != dims_fd3d_pml_kernel1_h[18][1] || xdim19 != dims_fd3d_pml_kernel1_h[19][0] || ydim19 != dims_fd3d_pml_kernel1_h[19][1] || xdim20 != dims_fd3d_pml_kernel1_h[20][0] || ydim20 != dims_fd3d_pml_kernel1_h[20][1] || xdim21 != dims_fd3d_pml_kernel1_h[21][0] || ydim21 != dims_fd3d_pml_kernel1_h[21][1] || xdim22 != dims_fd3d_pml_kernel1_h[22][0] || ydim22 != dims_fd3d_pml_kernel1_h[22][1] || xdim23 != dims_fd3d_pml_kernel1_h[23][0] || ydim23 != dims_fd3d_pml_kernel1_h[23][1] || xdim24 != dims_fd3d_pml_kernel1_h[24][0] || ydim24 != dims_fd3d_pml_kernel1_h[24][1] || xdim25 != dims_fd3d_pml_kernel1_h[25][0] || ydim25 != dims_fd3d_pml_kernel1_h[25][1] || xdim26 != dims_fd3d_pml_kernel1_h[26][0] || ydim26 != dims_fd3d_pml_kernel1_h[26][1]) {
        dims_fd3d_pml_kernel1_h[7][0] = xdim7;
        dims_fd3d_pml_kernel1_h[7][1] = ydim7;
        dims_fd3d_pml_kernel1_h[8][0] = xdim8;
        dims_fd3d_pml_kernel1_h[8][1] = ydim8;
        dims_fd3d_pml_kernel1_h[9][0] = xdim9;
        dims_fd3d_pml_kernel1_h[9][1] = ydim9;
        dims_fd3d_pml_kernel1_h[10][0] = xdim10;
        dims_fd3d_pml_kernel1_h[10][1] = ydim10;
        dims_fd3d_pml_kernel1_h[11][0] = xdim11;
        dims_fd3d_pml_kernel1_h[11][1] = ydim11;
        dims_fd3d_pml_kernel1_h[12][0] = xdim12;
        dims_fd3d_pml_kernel1_h[12][1] = ydim12;
        dims_fd3d_pml_kernel1_h[13][0] = xdim13;
        dims_fd3d_pml_kernel1_h[13][1] = ydim13;
        dims_fd3d_pml_kernel1_h[14][0] = xdim14;
        dims_fd3d_pml_kernel1_h[14][1] = ydim14;
        dims_fd3d_pml_kernel1_h[15][0] = xdim15;
        dims_fd3d_pml_kernel1_h[15][1] = ydim15;
        dims_fd3d_pml_kernel1_h[16][0] = xdim16;
        dims_fd3d_pml_kernel1_h[16][1] = ydim16;
        dims_fd3d_pml_kernel1_h[17][0] = xdim17;
        dims_fd3d_pml_kernel1_h[17][1] = ydim17;
        dims_fd3d_pml_kernel1_h[18][0] = xdim18;
        dims_fd3d_pml_kernel1_h[18][1] = ydim18;
        dims_fd3d_pml_kernel1_h[19][0] = xdim19;
        dims_fd3d_pml_kernel1_h[19][1] = ydim19;
        dims_fd3d_pml_kernel1_h[20][0] = xdim20;
        dims_fd3d_pml_kernel1_h[20][1] = ydim20;
        dims_fd3d_pml_kernel1_h[21][0] = xdim21;
        dims_fd3d_pml_kernel1_h[21][1] = ydim21;
        dims_fd3d_pml_kernel1_h[22][0] = xdim22;
        dims_fd3d_pml_kernel1_h[22][1] = ydim22;
        dims_fd3d_pml_kernel1_h[23][0] = xdim23;
        dims_fd3d_pml_kernel1_h[23][1] = ydim23;
        dims_fd3d_pml_kernel1_h[24][0] = xdim24;
        dims_fd3d_pml_kernel1_h[24][1] = ydim24;
        dims_fd3d_pml_kernel1_h[25][0] = xdim25;
        dims_fd3d_pml_kernel1_h[25][1] = ydim25;
        dims_fd3d_pml_kernel1_h[26][0] = xdim26;
        dims_fd3d_pml_kernel1_h[26][1] = ydim26;

        cutilSafeCall(block->instance->ostream(), hipMemcpyToSymbol(HIP_SYMBOL( dims_fd3d_pml_kernel1), dims_fd3d_pml_kernel1_h, sizeof(dims_fd3d_pml_kernel1)));
    }

    int x_size = MAX(0,end_indx[0]-start_indx[0]);
    int y_size = MAX(0,end_indx[1]-start_indx[1]);
    int z_size = MAX(0,end_indx[2]-start_indx[2]);

    dim3 grid( (x_size-1)/block->instance->OPS_block_size_x + 1, (y_size-1)/block->instance->OPS_block_size_y + 1, (z_size-1)/block->instance->OPS_block_size_z + 1);

    dim3 tblock(block->instance->OPS_block_size_x,block->instance->OPS_block_size_y,block->instance->OPS_block_size_z);

    long long int dat7 = (block->instance->OPS_soa ? args[7].dat->type_size : args[7].dat->elem_size);
    long long int dat8 = (block->instance->OPS_soa ? args[8].dat->type_size : args[8].dat->elem_size);
    long long int dat9 = (block->instance->OPS_soa ? args[9].dat->type_size : args[9].dat->elem_size);
    long long int dat10 = (block->instance->OPS_soa ? args[10].dat->type_size : args[10].dat->elem_size);
    long long int dat11 = (block->instance->OPS_soa ? args[11].dat->type_size : args[11].dat->elem_size);
    long long int dat12 = (block->instance->OPS_soa ? args[12].dat->type_size : args[12].dat->elem_size);
    long long int dat13 = (block->instance->OPS_soa ? args[13].dat->type_size : args[13].dat->elem_size);
    long long int dat14 = (block->instance->OPS_soa ? args[14].dat->type_size : args[14].dat->elem_size);
    long long int dat15 = (block->instance->OPS_soa ? args[15].dat->type_size : args[15].dat->elem_size);
    long long int dat16 = (block->instance->OPS_soa ? args[16].dat->type_size : args[16].dat->elem_size);
    long long int dat17 = (block->instance->OPS_soa ? args[17].dat->type_size : args[17].dat->elem_size);
    long long int dat18 = (block->instance->OPS_soa ? args[18].dat->type_size : args[18].dat->elem_size);
    long long int dat19 = (block->instance->OPS_soa ? args[19].dat->type_size : args[19].dat->elem_size);
    long long int dat20 = (block->instance->OPS_soa ? args[20].dat->type_size : args[20].dat->elem_size);
    long long int dat21 = (block->instance->OPS_soa ? args[21].dat->type_size : args[21].dat->elem_size);
    long long int dat22 = (block->instance->OPS_soa ? args[22].dat->type_size : args[22].dat->elem_size);
    long long int dat23 = (block->instance->OPS_soa ? args[23].dat->type_size : args[23].dat->elem_size);
    long long int dat24 = (block->instance->OPS_soa ? args[24].dat->type_size : args[24].dat->elem_size);
    long long int dat25 = (block->instance->OPS_soa ? args[25].dat->type_size : args[25].dat->elem_size);
    long long int dat26 = (block->instance->OPS_soa ? args[26].dat->type_size : args[26].dat->elem_size);

    char *p_a[27];

//  =======================
//  set up initial pointers
//  =======================
    long long int base7 = args[7].dat->base_offset + dat7 * 1 * (start_indx[0] * args[7].stencil->stride[0]);
    base7 = base7 + dat7 * 
                     args[7].dat->size[0] * 
                     (start_indx[1] * args[7].stencil->stride[1]);
    base7 = base7 + dat7 * 
                     args[7].dat->size[0] * 
                     args[7].dat->size[1] * 
                     (start_indx[2] * args[7].stencil->stride[2]);
    p_a[7] = (char *)args[7].data_d + base7;

    long long int base8 = args[8].dat->base_offset + dat8 * 1 * (start_indx[0] * args[8].stencil->stride[0]);
    base8 = base8 + dat8 * 
                     args[8].dat->size[0] * 
                     (start_indx[1] * args[8].stencil->stride[1]);
    base8 = base8 + dat8 * 
                     args[8].dat->size[0] * 
                     args[8].dat->size[1] * 
                     (start_indx[2] * args[8].stencil->stride[2]);
    p_a[8] = (char *)args[8].data_d + base8;

    long long int base9 = args[9].dat->base_offset + dat9 * 1 * (start_indx[0] * args[9].stencil->stride[0]);
    base9 = base9 + dat9 * 
                     args[9].dat->size[0] * 
                     (start_indx[1] * args[9].stencil->stride[1]);
    base9 = base9 + dat9 * 
                     args[9].dat->size[0] * 
                     args[9].dat->size[1] * 
                     (start_indx[2] * args[9].stencil->stride[2]);
    p_a[9] = (char *)args[9].data_d + base9;

    long long int base10 = args[10].dat->base_offset + dat10 * 1 * (start_indx[0] * args[10].stencil->stride[0]);
    base10 = base10 + dat10 * 
                     args[10].dat->size[0] * 
                     (start_indx[1] * args[10].stencil->stride[1]);
    base10 = base10 + dat10 * 
                     args[10].dat->size[0] * 
                     args[10].dat->size[1] * 
                     (start_indx[2] * args[10].stencil->stride[2]);
    p_a[10] = (char *)args[10].data_d + base10;

    long long int base11 = args[11].dat->base_offset + dat11 * 1 * (start_indx[0] * args[11].stencil->stride[0]);
    base11 = base11 + dat11 * 
                     args[11].dat->size[0] * 
                     (start_indx[1] * args[11].stencil->stride[1]);
    base11 = base11 + dat11 * 
                     args[11].dat->size[0] * 
                     args[11].dat->size[1] * 
                     (start_indx[2] * args[11].stencil->stride[2]);
    p_a[11] = (char *)args[11].data_d + base11;

    long long int base12 = args[12].dat->base_offset + dat12 * 1 * (start_indx[0] * args[12].stencil->stride[0]);
    base12 = base12 + dat12 * 
                     args[12].dat->size[0] * 
                     (start_indx[1] * args[12].stencil->stride[1]);
    base12 = base12 + dat12 * 
                     args[12].dat->size[0] * 
                     args[12].dat->size[1] * 
                     (start_indx[2] * args[12].stencil->stride[2]);
    p_a[12] = (char *)args[12].data_d + base12;

    long long int base13 = args[13].dat->base_offset + dat13 * 1 * (start_indx[0] * args[13].stencil->stride[0]);
    base13 = base13 + dat13 * 
                     args[13].dat->size[0] * 
                     (start_indx[1] * args[13].stencil->stride[1]);
    base13 = base13 + dat13 * 
                     args[13].dat->size[0] * 
                     args[13].dat->size[1] * 
                     (start_indx[2] * args[13].stencil->stride[2]);
    p_a[13] = (char *)args[13].data_d + base13;

    long long int base14 = args[14].dat->base_offset + dat14 * 1 * (start_indx[0] * args[14].stencil->stride[0]);
    base14 = base14 + dat14 * 
                     args[14].dat->size[0] * 
                     (start_indx[1] * args[14].stencil->stride[1]);
    base14 = base14 + dat14 * 
                     args[14].dat->size[0] * 
                     args[14].dat->size[1] * 
                     (start_indx[2] * args[14].stencil->stride[2]);
    p_a[14] = (char *)args[14].data_d + base14;

    long long int base15 = args[15].dat->base_offset + dat15 * 1 * (start_indx[0] * args[15].stencil->stride[0]);
    base15 = base15 + dat15 * 
                     args[15].dat->size[0] * 
                     (start_indx[1] * args[15].stencil->stride[1]);
    base15 = base15 + dat15 * 
                     args[15].dat->size[0] * 
                     args[15].dat->size[1] * 
                     (start_indx[2] * args[15].stencil->stride[2]);
    p_a[15] = (char *)args[15].data_d + base15;

    long long int base16 = args[16].dat->base_offset + dat16 * 1 * (start_indx[0] * args[16].stencil->stride[0]);
    base16 = base16 + dat16 * 
                     args[16].dat->size[0] * 
                     (start_indx[1] * args[16].stencil->stride[1]);
    base16 = base16 + dat16 * 
                     args[16].dat->size[0] * 
                     args[16].dat->size[1] * 
                     (start_indx[2] * args[16].stencil->stride[2]);
    p_a[16] = (char *)args[16].data_d + base16;

    long long int base17 = args[17].dat->base_offset + dat17 * 1 * (start_indx[0] * args[17].stencil->stride[0]);
    base17 = base17 + dat17 * 
                     args[17].dat->size[0] * 
                     (start_indx[1] * args[17].stencil->stride[1]);
    base17 = base17 + dat17 * 
                     args[17].dat->size[0] * 
                     args[17].dat->size[1] * 
                     (start_indx[2] * args[17].stencil->stride[2]);
    p_a[17] = (char *)args[17].data_d + base17;

    long long int base18 = args[18].dat->base_offset + dat18 * 1 * (start_indx[0] * args[18].stencil->stride[0]);
    base18 = base18 + dat18 * 
                     args[18].dat->size[0] * 
                     (start_indx[1] * args[18].stencil->stride[1]);
    base18 = base18 + dat18 * 
                     args[18].dat->size[0] * 
                     args[18].dat->size[1] * 
                     (start_indx[2] * args[18].stencil->stride[2]);
    p_a[18] = (char *)args[18].data_d + base18;

    long long int base19 = args[19].dat->base_offset + dat19 * 1 * (start_indx[0] * args[19].stencil->stride[0]);
    base19 = base19 + dat19 * 
                     args[19].dat->size[0] * 
                     (start_indx[1] * args[19].stencil->stride[1]);
    base19 = base19 + dat19 * 
                     args[19].dat->size[0] * 
                     args[19].dat->size[1] * 
                     (start_indx[2] * args[19].stencil->stride[2]);
    p_a[19] = (char *)args[19].data_d + base19;

    long long int base20 = args[20].dat->base_offset + dat20 * 1 * (start_indx[0] * args[20].stencil->stride[0]);
    base20 = base20 + dat20 * 
                     args[20].dat->size[0] * 
                     (start_indx[1] * args[20].stencil->stride[1]);
    base20 = base20 + dat20 * 
                     args[20].dat->size[0] * 
                     args[20].dat->size[1] * 
                     (start_indx[2] * args[20].stencil->stride[2]);
    p_a[20] = (char *)args[20].data_d + base20;

    long long int base21 = args[21].dat->base_offset + dat21 * 1 * (start_indx[0] * args[21].stencil->stride[0]);
    base21 = base21 + dat21 * 
                     args[21].dat->size[0] * 
                     (start_indx[1] * args[21].stencil->stride[1]);
    base21 = base21 + dat21 * 
                     args[21].dat->size[0] * 
                     args[21].dat->size[1] * 
                     (start_indx[2] * args[21].stencil->stride[2]);
    p_a[21] = (char *)args[21].data_d + base21;

    long long int base22 = args[22].dat->base_offset + dat22 * 1 * (start_indx[0] * args[22].stencil->stride[0]);
    base22 = base22 + dat22 * 
                     args[22].dat->size[0] * 
                     (start_indx[1] * args[22].stencil->stride[1]);
    base22 = base22 + dat22 * 
                     args[22].dat->size[0] * 
                     args[22].dat->size[1] * 
                     (start_indx[2] * args[22].stencil->stride[2]);
    p_a[22] = (char *)args[22].data_d + base22;

    long long int base23 = args[23].dat->base_offset + dat23 * 1 * (start_indx[0] * args[23].stencil->stride[0]);
    base23 = base23 + dat23 * 
                     args[23].dat->size[0] * 
                     (start_indx[1] * args[23].stencil->stride[1]);
    base23 = base23 + dat23 * 
                     args[23].dat->size[0] * 
                     args[23].dat->size[1] * 
                     (start_indx[2] * args[23].stencil->stride[2]);
    p_a[23] = (char *)args[23].data_d + base23;

    long long int base24 = args[24].dat->base_offset + dat24 * 1 * (start_indx[0] * args[24].stencil->stride[0]);
    base24 = base24 + dat24 * 
                     args[24].dat->size[0] * 
                     (start_indx[1] * args[24].stencil->stride[1]);
    base24 = base24 + dat24 * 
                     args[24].dat->size[0] * 
                     args[24].dat->size[1] * 
                     (start_indx[2] * args[24].stencil->stride[2]);
    p_a[24] = (char *)args[24].data_d + base24;

    long long int base25 = args[25].dat->base_offset + dat25 * 1 * (start_indx[0] * args[25].stencil->stride[0]);
    base25 = base25 + dat25 * 
                     args[25].dat->size[0] * 
                     (start_indx[1] * args[25].stencil->stride[1]);
    base25 = base25 + dat25 * 
                     args[25].dat->size[0] * 
                     args[25].dat->size[1] * 
                     (start_indx[2] * args[25].stencil->stride[2]);
    p_a[25] = (char *)args[25].data_d + base25;

    long long int base26 = args[26].dat->base_offset + dat26 * 1 * (start_indx[0] * args[26].stencil->stride[0]);
    base26 = base26 + dat26 * 
                     args[26].dat->size[0] * 
                     (start_indx[1] * args[26].stencil->stride[1]);
    base26 = base26 + dat26 * 
                     args[26].dat->size[0] * 
                     args[26].dat->size[1] * 
                     (start_indx[2] * args[26].stencil->stride[2]);
    p_a[26] = (char *)args[26].data_d + base26;

//  =============
//  Halo exchange
//  =============
#ifndef OPS_LAZY
    ops_H_D_exchanges_device(args, 27);
    ops_halo_exchanges(args, 27, range);
#endif

    if (block->instance->OPS_diags > 1) { 
        ops_timers_core(&__c2, &__t2);
        block->instance->OPS_kernels[3].mpi_time += __t2 - __t1;
    }

//  ==========================================================
//  ops_dat strides for offset calculation in wrapper function
//  ==========================================================
    int xstride_7, ystride_7, zstride_7;
    xstride_7 = args[7].stencil->stride[0];    ystride_7 = args[7].stencil->stride[1];
    zstride_7 = args[7].stencil->stride[2];
    int xstride_8, ystride_8, zstride_8;
    xstride_8 = args[8].stencil->stride[0];    ystride_8 = args[8].stencil->stride[1];
    zstride_8 = args[8].stencil->stride[2];
    int xstride_9, ystride_9, zstride_9;
    xstride_9 = args[9].stencil->stride[0];    ystride_9 = args[9].stencil->stride[1];
    zstride_9 = args[9].stencil->stride[2];
    int xstride_10, ystride_10, zstride_10;
    xstride_10 = args[10].stencil->stride[0];    ystride_10 = args[10].stencil->stride[1];
    zstride_10 = args[10].stencil->stride[2];
    int xstride_11, ystride_11, zstride_11;
    xstride_11 = args[11].stencil->stride[0];    ystride_11 = args[11].stencil->stride[1];
    zstride_11 = args[11].stencil->stride[2];
    int xstride_12, ystride_12, zstride_12;
    xstride_12 = args[12].stencil->stride[0];    ystride_12 = args[12].stencil->stride[1];
    zstride_12 = args[12].stencil->stride[2];
    int xstride_13, ystride_13, zstride_13;
    xstride_13 = args[13].stencil->stride[0];    ystride_13 = args[13].stencil->stride[1];
    zstride_13 = args[13].stencil->stride[2];
    int xstride_14, ystride_14, zstride_14;
    xstride_14 = args[14].stencil->stride[0];    ystride_14 = args[14].stencil->stride[1];
    zstride_14 = args[14].stencil->stride[2];
    int xstride_15, ystride_15, zstride_15;
    xstride_15 = args[15].stencil->stride[0];    ystride_15 = args[15].stencil->stride[1];
    zstride_15 = args[15].stencil->stride[2];
    int xstride_16, ystride_16, zstride_16;
    xstride_16 = args[16].stencil->stride[0];    ystride_16 = args[16].stencil->stride[1];
    zstride_16 = args[16].stencil->stride[2];
    int xstride_17, ystride_17, zstride_17;
    xstride_17 = args[17].stencil->stride[0];    ystride_17 = args[17].stencil->stride[1];
    zstride_17 = args[17].stencil->stride[2];
    int xstride_18, ystride_18, zstride_18;
    xstride_18 = args[18].stencil->stride[0];    ystride_18 = args[18].stencil->stride[1];
    zstride_18 = args[18].stencil->stride[2];
    int xstride_19, ystride_19, zstride_19;
    xstride_19 = args[19].stencil->stride[0];    ystride_19 = args[19].stencil->stride[1];
    zstride_19 = args[19].stencil->stride[2];
    int xstride_20, ystride_20, zstride_20;
    xstride_20 = args[20].stencil->stride[0];    ystride_20 = args[20].stencil->stride[1];
    zstride_20 = args[20].stencil->stride[2];
    int xstride_21, ystride_21, zstride_21;
    xstride_21 = args[21].stencil->stride[0];    ystride_21 = args[21].stencil->stride[1];
    zstride_21 = args[21].stencil->stride[2];
    int xstride_22, ystride_22, zstride_22;
    xstride_22 = args[22].stencil->stride[0];    ystride_22 = args[22].stencil->stride[1];
    zstride_22 = args[22].stencil->stride[2];
    int xstride_23, ystride_23, zstride_23;
    xstride_23 = args[23].stencil->stride[0];    ystride_23 = args[23].stencil->stride[1];
    zstride_23 = args[23].stencil->stride[2];
    int xstride_24, ystride_24, zstride_24;
    xstride_24 = args[24].stencil->stride[0];    ystride_24 = args[24].stencil->stride[1];
    zstride_24 = args[24].stencil->stride[2];
    int xstride_25, ystride_25, zstride_25;
    xstride_25 = args[25].stencil->stride[0];    ystride_25 = args[25].stencil->stride[1];
    zstride_25 = args[25].stencil->stride[2];
    int xstride_26, ystride_26, zstride_26;
    xstride_26 = args[26].stencil->stride[0];    ystride_26 = args[26].stencil->stride[1];
    zstride_26 = args[26].stencil->stride[2];

//  call kernel wrapper function, passing in pointers to data
    if (x_size > 0 && y_size > 0 && z_size > 0) {

        ops_fd3d_pml_kernel1<<<grid, tblock >>> (
                   *(int *)arg0.data, 
                   *(int *)arg1.data, 
                   *(int *)arg2.data, 
                   arg_idx[0], arg_idx[1], arg_idx[2], 
                   *(float *)arg4.data, 
                   *(float *)arg5.data, 
                   *(float *)arg6.data, 
                   (float *)p_a[7], xstride_7, ystride_7, zstride_7, 
                   (float *)p_a[8], xstride_8, ystride_8, zstride_8, 
                   (float *)p_a[9], xstride_9, ystride_9, zstride_9, 
                   (float *)p_a[10], xstride_10, ystride_10, zstride_10, 
                   (float *)p_a[11], xstride_11, ystride_11, zstride_11, 
                   (float *)p_a[12], xstride_12, ystride_12, zstride_12, 
                   (float *)p_a[13], xstride_13, ystride_13, zstride_13, 
                   (float *)p_a[14], xstride_14, ystride_14, zstride_14, 
                   (float *)p_a[15], xstride_15, ystride_15, zstride_15, 
                   (float *)p_a[16], xstride_16, ystride_16, zstride_16, 
                   (float *)p_a[17], xstride_17, ystride_17, zstride_17, 
                   (float *)p_a[18], xstride_18, ystride_18, zstride_18, 
                   (float *)p_a[19], xstride_19, ystride_19, zstride_19, 
                   (float *)p_a[20], xstride_20, ystride_20, zstride_20, 
                   (float *)p_a[21], xstride_21, ystride_21, zstride_21, 
                   (float *)p_a[22], xstride_22, ystride_22, zstride_22, 
                   (float *)p_a[23], xstride_23, ystride_23, zstride_23, 
                   (float *)p_a[24], xstride_24, ystride_24, zstride_24, 
                   (float *)p_a[25], xstride_25, ystride_25, zstride_25, 
                   (float *)p_a[26], xstride_26, ystride_26, zstride_26, 
                   x_size, y_size, z_size);

    }

    cutilSafeCall(block->instance->ostream(), hipGetLastError());

    if(block->instance->OPS_diags > 1) {
        cutilSafeCall(block->instance->ostream(), hipDeviceSynchronize());
        ops_timers_core(&__c1, &__t1);
        block->instance->OPS_kernels[3].time += __t1 - __t2;
    }

#ifndef OPS_LAZY
    ops_set_dirtybit_device(args, 27);
    ops_set_halo_dirtybit3(&args[15], range);
    ops_set_halo_dirtybit3(&args[16], range);
    ops_set_halo_dirtybit3(&args[17], range);
    ops_set_halo_dirtybit3(&args[18], range);
    ops_set_halo_dirtybit3(&args[19], range);
    ops_set_halo_dirtybit3(&args[20], range);
    ops_set_halo_dirtybit3(&args[21], range);
    ops_set_halo_dirtybit3(&args[22], range);
    ops_set_halo_dirtybit3(&args[23], range);
    ops_set_halo_dirtybit3(&args[24], range);
    ops_set_halo_dirtybit3(&args[25], range);
    ops_set_halo_dirtybit3(&args[26], range);
#endif

    if (block->instance->OPS_diags > 1) {
//      ====================
//      Update kernel record
//      ====================
        ops_timers_core(&__c2, &__t2);
        block->instance->OPS_kernels[3].mpi_time += __t2 - __t1;
        block->instance->OPS_kernels[3].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg7);
        block->instance->OPS_kernels[3].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg8);
        block->instance->OPS_kernels[3].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg9);
        block->instance->OPS_kernels[3].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg10);
        block->instance->OPS_kernels[3].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg11);
        block->instance->OPS_kernels[3].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg12);
        block->instance->OPS_kernels[3].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg13);
        block->instance->OPS_kernels[3].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg14);
        block->instance->OPS_kernels[3].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg15);
        block->instance->OPS_kernels[3].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg16);
        block->instance->OPS_kernels[3].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg17);
        block->instance->OPS_kernels[3].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg18);
        block->instance->OPS_kernels[3].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg19);
        block->instance->OPS_kernels[3].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg20);
        block->instance->OPS_kernels[3].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg21);
        block->instance->OPS_kernels[3].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg22);
        block->instance->OPS_kernels[3].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg23);
        block->instance->OPS_kernels[3].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg24);
        block->instance->OPS_kernels[3].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg25);
        block->instance->OPS_kernels[3].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg26);
    }
}

#ifdef OPS_LAZY
void ops_par_loop_fd3d_pml_kernel1(
    const char * name,
    ops_block block,
    int dim,
    int * range,
    ops_arg arg0,
    ops_arg arg1,
    ops_arg arg2,
    ops_arg arg3,
    ops_arg arg4,
    ops_arg arg5,
    ops_arg arg6,
    ops_arg arg7,
    ops_arg arg8,
    ops_arg arg9,
    ops_arg arg10,
    ops_arg arg11,
    ops_arg arg12,
    ops_arg arg13,
    ops_arg arg14,
    ops_arg arg15,
    ops_arg arg16,
    ops_arg arg17,
    ops_arg arg18,
    ops_arg arg19,
    ops_arg arg20,
    ops_arg arg21,
    ops_arg arg22,
    ops_arg arg23,
    ops_arg arg24,
    ops_arg arg25,
    ops_arg arg26
    )
{
    ops_arg args[27];

    args[0] = arg0;
    args[1] = arg1;
    args[2] = arg2;
    args[3] = arg3;
    args[4] = arg4;
    args[5] = arg5;
    args[6] = arg6;
    args[7] = arg7;
    args[8] = arg8;
    args[9] = arg9;
    args[10] = arg10;
    args[11] = arg11;
    args[12] = arg12;
    args[13] = arg13;
    args[14] = arg14;
    args[15] = arg15;
    args[16] = arg16;
    args[17] = arg17;
    args[18] = arg18;
    args[19] = arg19;
    args[20] = arg20;
    args[21] = arg21;
    args[22] = arg22;
    args[23] = arg23;
    args[24] = arg24;
    args[25] = arg25;
    args[26] = arg26;

    create_kerneldesc_and_enque(name, "fd3d_pml_kernel1", args, 27, 3, dim, 1, range, block, ops_par_loop_fd3d_pml_kernel1_execute);
}
#endif
