#include "hip/hip_runtime.h"
// Auto-generated at 2025-08-06 23:23:48.183093 by ops-translator

__constant__ int dims_rtm_kernel_populate[7][2];
static int dims_rtm_kernel_populate_h[7][2] = {{0}};

//  =============
//  User function
//  =============
__device__ void rtm_kernel_populate_gpu(const int *dispx, const int *dispy, const int *dispz, const int *idx, ACC<float> &rho, ACC<float> &mu, ACC<float> &yy_0) {

    float x = 1.0*((float)(idx[0]-nx/2)/nx);
    float y = 1.0*((float)(idx[1]-ny/2)/ny);
    float z = 1.0*((float)(idx[2]-nz/2)/nz);

    const float C = 1.0f;
    const float r0 = 0.001f;
    rho(0,0,0) = 1000.0f; 
    mu(0,0,0) = 0.001f; 

    yy_0(0,0,0) = (1./3.)*C*exp(-(x*x+y*y+z*z)/r0); 

}

//  ============================
//  Cuda kernel wrapper function
//  ============================
__global__ void ops_rtm_kernel_populate(const int arg0, 
const int arg1, 
const int arg2, 
int arg_idx0, int arg_idx1, int arg_idx2, 
float* __restrict arg4, int xstride_4, int ystride_4, int zstride_4, 
float* __restrict arg5, int xstride_5, int ystride_5, int zstride_5, 
float* __restrict arg6, int xstride_6, int ystride_6, int zstride_6, 
int size0, int size1, int size2) {

    int idx_z = blockDim.z * blockIdx.z + threadIdx.z;
    int idx_y = blockDim.y * blockIdx.y + threadIdx.y;
    int idx_x = blockDim.x * blockIdx.x + threadIdx.x;

    int arg_idx[3];
    arg_idx[0] = arg_idx0+idx_x;
    arg_idx[1] = arg_idx1+idx_y;
    arg_idx[2] = arg_idx2+idx_z;

    arg4 += idx_x * xstride_4*1 + idx_y * ystride_4*1 * dims_rtm_kernel_populate[4][0] + idx_z * zstride_4*1 * dims_rtm_kernel_populate[4][0] * dims_rtm_kernel_populate[4][1];
    arg5 += idx_x * xstride_5*1 + idx_y * ystride_5*1 * dims_rtm_kernel_populate[5][0] + idx_z * zstride_5*1 * dims_rtm_kernel_populate[5][0] * dims_rtm_kernel_populate[5][1];
    arg6 += idx_x * xstride_6*1 + idx_y * ystride_6*1 * dims_rtm_kernel_populate[6][0] + idx_z * zstride_6*1 * dims_rtm_kernel_populate[6][0] * dims_rtm_kernel_populate[6][1];

    if(idx_x < size0 && idx_y < size1 && idx_z < size2) {

        ACC<float> argp4(dims_rtm_kernel_populate[4][0], dims_rtm_kernel_populate[4][1], arg4);
        ACC<float> argp5(dims_rtm_kernel_populate[5][0], dims_rtm_kernel_populate[5][1], arg5);
        ACC<float> argp6(dims_rtm_kernel_populate[6][0], dims_rtm_kernel_populate[6][1], arg6);

        rtm_kernel_populate_gpu(&arg0, &arg1, &arg2, arg_idx, argp4, argp5, argp6);

    }// End of cuda index in_range check

}// End of cuda kernel wrapper function

//  ==================
//  Host stub function
//  ==================
#ifndef OPS_LAZY
void ops_par_loop_rtm_kernel_populate(
    const char * name,
    ops_block block,
    int dim,
    int *range,
    ops_arg arg0,
    ops_arg arg1,
    ops_arg arg2,
    ops_arg arg3,
    ops_arg arg4,
    ops_arg arg5,
    ops_arg arg6
)
{ 
#else
void ops_par_loop_rtm_kernel_populate_execute(ops_kernel_descriptor *desc)
{
    ops_block block = desc->block;
    int dim = desc->dim;
    int *range = desc->range;
    ops_arg arg0 = desc->args[0];
    ops_arg arg1 = desc->args[1];
    ops_arg arg2 = desc->args[2];
    ops_arg arg3 = desc->args[3];
    ops_arg arg4 = desc->args[4];
    ops_arg arg5 = desc->args[5];
    ops_arg arg6 = desc->args[6];
#endif

//  ======
//  Timing
//  ======
    double __t1, __t2, __c1, __c2;

    ops_arg args[7];

    args[0] = arg0;
    args[1] = arg1;
    args[2] = arg2;
    args[3] = arg3;
    args[4] = arg4;
    args[5] = arg5;
    args[6] = arg6;

#if defined(CHECKPOINTING) && !defined(OPS_LAZY)
    if (!ops_checkpointing_before(args, 7, range, 1)) return;
#endif

    if (block->instance->OPS_diags > 1)
    {
        ops_timing_realloc(block->instance, 1, "rtm_kernel_populate");
        block->instance->OPS_kernels[1].count++;
        ops_timers_core(&__c1, &__t1);
    }

//  =================================================
//  compute locally allocated range for the sub-block
//  =================================================
    int start_indx[3];
    int end_indx[3];
    int arg_idx[3];

#if defined(OPS_LAZY) || !defined(OPS_MPI)
    for (int n = 0; n < 3; n++) {
        start_indx[n] = range[2*n];
        end_indx[n]   = range[2*n+1];
    }
#else
    if (compute_ranges(args, 7, block, range, start_indx, end_indx, arg_idx) < 0) return;
#endif

#if defined(OPS_MPI)
#if defined(OPS_LAZY)
    sub_block_list sb = OPS_sub_block_list[block->index];
    arg_idx[0] = sb->decomp_disp[0]+start_indx[0];
    arg_idx[1] = sb->decomp_disp[1]+start_indx[1];
    arg_idx[2] = sb->decomp_disp[2]+start_indx[2];
#endif  // OPS_LAZY
#else   // OPS_MPI
    arg_idx[0] = start_indx[0];
    arg_idx[1] = start_indx[1];
    arg_idx[2] = start_indx[2];
#endif  // OPS_MPI

    int xdim4 = args[4].dat->size[0];
    int ydim4 = args[4].dat->size[1];
    int xdim5 = args[5].dat->size[0];
    int ydim5 = args[5].dat->size[1];
    int xdim6 = args[6].dat->size[0];
    int ydim6 = args[6].dat->size[1];

    if (xdim4 != dims_rtm_kernel_populate_h[4][0] || ydim4 != dims_rtm_kernel_populate_h[4][1] || xdim5 != dims_rtm_kernel_populate_h[5][0] || ydim5 != dims_rtm_kernel_populate_h[5][1] || xdim6 != dims_rtm_kernel_populate_h[6][0] || ydim6 != dims_rtm_kernel_populate_h[6][1]) {
        dims_rtm_kernel_populate_h[4][0] = xdim4;
        dims_rtm_kernel_populate_h[4][1] = ydim4;
        dims_rtm_kernel_populate_h[5][0] = xdim5;
        dims_rtm_kernel_populate_h[5][1] = ydim5;
        dims_rtm_kernel_populate_h[6][0] = xdim6;
        dims_rtm_kernel_populate_h[6][1] = ydim6;

        cutilSafeCall(block->instance->ostream(), hipMemcpyToSymbol(HIP_SYMBOL( dims_rtm_kernel_populate), dims_rtm_kernel_populate_h, sizeof(dims_rtm_kernel_populate)));
    }

    int x_size = MAX(0,end_indx[0]-start_indx[0]);
    int y_size = MAX(0,end_indx[1]-start_indx[1]);
    int z_size = MAX(0,end_indx[2]-start_indx[2]);

    dim3 grid( (x_size-1)/block->instance->OPS_block_size_x + 1, (y_size-1)/block->instance->OPS_block_size_y + 1, (z_size-1)/block->instance->OPS_block_size_z + 1);

    dim3 tblock(block->instance->OPS_block_size_x,block->instance->OPS_block_size_y,block->instance->OPS_block_size_z);

    long long int dat4 = (block->instance->OPS_soa ? args[4].dat->type_size : args[4].dat->elem_size);
    long long int dat5 = (block->instance->OPS_soa ? args[5].dat->type_size : args[5].dat->elem_size);
    long long int dat6 = (block->instance->OPS_soa ? args[6].dat->type_size : args[6].dat->elem_size);

    char *p_a[7];

//  =======================
//  set up initial pointers
//  =======================
    long long int base4 = args[4].dat->base_offset + dat4 * 1 * (start_indx[0] * args[4].stencil->stride[0]);
    base4 = base4 + dat4 * 
                     args[4].dat->size[0] * 
                     (start_indx[1] * args[4].stencil->stride[1]);
    base4 = base4 + dat4 * 
                     args[4].dat->size[0] * 
                     args[4].dat->size[1] * 
                     (start_indx[2] * args[4].stencil->stride[2]);
    p_a[4] = (char *)args[4].data_d + base4;

    long long int base5 = args[5].dat->base_offset + dat5 * 1 * (start_indx[0] * args[5].stencil->stride[0]);
    base5 = base5 + dat5 * 
                     args[5].dat->size[0] * 
                     (start_indx[1] * args[5].stencil->stride[1]);
    base5 = base5 + dat5 * 
                     args[5].dat->size[0] * 
                     args[5].dat->size[1] * 
                     (start_indx[2] * args[5].stencil->stride[2]);
    p_a[5] = (char *)args[5].data_d + base5;

    long long int base6 = args[6].dat->base_offset + dat6 * 1 * (start_indx[0] * args[6].stencil->stride[0]);
    base6 = base6 + dat6 * 
                     args[6].dat->size[0] * 
                     (start_indx[1] * args[6].stencil->stride[1]);
    base6 = base6 + dat6 * 
                     args[6].dat->size[0] * 
                     args[6].dat->size[1] * 
                     (start_indx[2] * args[6].stencil->stride[2]);
    p_a[6] = (char *)args[6].data_d + base6;

//  =============
//  Halo exchange
//  =============
#ifndef OPS_LAZY
    ops_H_D_exchanges_device(args, 7);
    ops_halo_exchanges(args, 7, range);
#endif

    if (block->instance->OPS_diags > 1) { 
        ops_timers_core(&__c2, &__t2);
        block->instance->OPS_kernels[1].mpi_time += __t2 - __t1;
    }

//  ==========================================================
//  ops_dat strides for offset calculation in wrapper function
//  ==========================================================
    int xstride_4, ystride_4, zstride_4;
    xstride_4 = args[4].stencil->stride[0];    ystride_4 = args[4].stencil->stride[1];
    zstride_4 = args[4].stencil->stride[2];
    int xstride_5, ystride_5, zstride_5;
    xstride_5 = args[5].stencil->stride[0];    ystride_5 = args[5].stencil->stride[1];
    zstride_5 = args[5].stencil->stride[2];
    int xstride_6, ystride_6, zstride_6;
    xstride_6 = args[6].stencil->stride[0];    ystride_6 = args[6].stencil->stride[1];
    zstride_6 = args[6].stencil->stride[2];

//  call kernel wrapper function, passing in pointers to data
    if (x_size > 0 && y_size > 0 && z_size > 0) {

        ops_rtm_kernel_populate<<<grid, tblock >>> (
                   *(int *)arg0.data, 
                   *(int *)arg1.data, 
                   *(int *)arg2.data, 
                   arg_idx[0], arg_idx[1], arg_idx[2], 
                   (float *)p_a[4], xstride_4, ystride_4, zstride_4, 
                   (float *)p_a[5], xstride_5, ystride_5, zstride_5, 
                   (float *)p_a[6], xstride_6, ystride_6, zstride_6, 
                   x_size, y_size, z_size);

    }

    cutilSafeCall(block->instance->ostream(), hipGetLastError());

    if(block->instance->OPS_diags > 1) {
        cutilSafeCall(block->instance->ostream(), hipDeviceSynchronize());
        ops_timers_core(&__c1, &__t1);
        block->instance->OPS_kernels[1].time += __t1 - __t2;
    }

#ifndef OPS_LAZY
    ops_set_dirtybit_device(args, 7);
    ops_set_halo_dirtybit3(&args[4], range);
    ops_set_halo_dirtybit3(&args[5], range);
    ops_set_halo_dirtybit3(&args[6], range);
#endif

    if (block->instance->OPS_diags > 1) {
//      ====================
//      Update kernel record
//      ====================
        ops_timers_core(&__c2, &__t2);
        block->instance->OPS_kernels[1].mpi_time += __t2 - __t1;
        block->instance->OPS_kernels[1].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg4);
        block->instance->OPS_kernels[1].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg5);
        block->instance->OPS_kernels[1].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg6);
    }
}

#ifdef OPS_LAZY
void ops_par_loop_rtm_kernel_populate(
    const char * name,
    ops_block block,
    int dim,
    int * range,
    ops_arg arg0,
    ops_arg arg1,
    ops_arg arg2,
    ops_arg arg3,
    ops_arg arg4,
    ops_arg arg5,
    ops_arg arg6
    )
{
    ops_arg args[7];

    args[0] = arg0;
    args[1] = arg1;
    args[2] = arg2;
    args[3] = arg3;
    args[4] = arg4;
    args[5] = arg5;
    args[6] = arg6;

    create_kerneldesc_and_enque(name, "rtm_kernel_populate", args, 7, 1, dim, 1, range, block, ops_par_loop_rtm_kernel_populate_execute);
}
#endif
