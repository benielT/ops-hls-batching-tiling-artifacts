// Auto-generated at 2025-08-06 23:23:48.476355 by ops-translator

// headers
#define OPS_3D
#define OPS_API 2
#include "ops_lib_core.h"

#ifdef OPS_MPI
#include "ops_mpi_core.h"
#include <limits>
#endif

#include "ops_cuda_rt_support.h"
#include "ops_cuda_reduction.h"

#include <hip/hip_complex.h>

#define OPS_FUN_PREFIX __device__ __host__
#include "user_types.h"

//  global constants
__constant__ float dx;
__constant__ float dy;
__constant__ float dz;
__constant__ float invdx;
__constant__ float invdy;
__constant__ float invdz;
__constant__ int nx;
__constant__ int ny;
__constant__ int nz;
__constant__ int pml_width;
__constant__ int half_order;
__constant__ int order;

void ops_init_backend(){}

void ops_decl_const_char(OPS_instance *instance, int dim, char const *type, int size, char *dat, char const *name) {
    ops_execute(instance);

    if(!strcmp(name, "dx")) {
        cutilSafeCall(instance->ostream(), hipMemcpyToSymbol(HIP_SYMBOL(dx), dat, dim*size));
    } 
    else
    if(!strcmp(name, "dy")) {
        cutilSafeCall(instance->ostream(), hipMemcpyToSymbol(HIP_SYMBOL(dy), dat, dim*size));
    } 
    else
    if(!strcmp(name, "dz")) {
        cutilSafeCall(instance->ostream(), hipMemcpyToSymbol(HIP_SYMBOL(dz), dat, dim*size));
    } 
    else
    if(!strcmp(name, "invdx")) {
        cutilSafeCall(instance->ostream(), hipMemcpyToSymbol(HIP_SYMBOL(invdx), dat, dim*size));
    } 
    else
    if(!strcmp(name, "invdy")) {
        cutilSafeCall(instance->ostream(), hipMemcpyToSymbol(HIP_SYMBOL(invdy), dat, dim*size));
    } 
    else
    if(!strcmp(name, "invdz")) {
        cutilSafeCall(instance->ostream(), hipMemcpyToSymbol(HIP_SYMBOL(invdz), dat, dim*size));
    } 
    else
    if(!strcmp(name, "nx")) {
        cutilSafeCall(instance->ostream(), hipMemcpyToSymbol(HIP_SYMBOL(nx), dat, dim*size));
    } 
    else
    if(!strcmp(name, "ny")) {
        cutilSafeCall(instance->ostream(), hipMemcpyToSymbol(HIP_SYMBOL(ny), dat, dim*size));
    } 
    else
    if(!strcmp(name, "nz")) {
        cutilSafeCall(instance->ostream(), hipMemcpyToSymbol(HIP_SYMBOL(nz), dat, dim*size));
    } 
    else
    if(!strcmp(name, "pml_width")) {
        cutilSafeCall(instance->ostream(), hipMemcpyToSymbol(HIP_SYMBOL(pml_width), dat, dim*size));
    } 
    else
    if(!strcmp(name, "half_order")) {
        cutilSafeCall(instance->ostream(), hipMemcpyToSymbol(HIP_SYMBOL(half_order), dat, dim*size));
    } 
    else
    if(!strcmp(name, "order")) {
        cutilSafeCall(instance->ostream(), hipMemcpyToSymbol(HIP_SYMBOL(order), dat, dim*size));
    } 
    else
    {
        throw OPSException(OPS_RUNTIME_ERROR, "error: unknown const name");
    }
}

// user kernel files
#include "rtm_kernel_populate_kernel.cu"
#include "kernel_copy_kernel.cu"
#include "fd3d_pml_kernel1_kernel.cu"
#include "fd3d_pml_kernel2_kernel.cu"
#include "fd3d_pml_kernel3_kernel.cu"

